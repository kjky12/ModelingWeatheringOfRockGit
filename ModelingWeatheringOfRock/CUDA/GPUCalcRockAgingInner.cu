#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"

#define dfSHARED_MEMORY_SIZE	32*7

#define dfCALC_BLOCK_CNT	8


__device__ __host__ int gcd(int a, int b)
{
	int c;
	while (b != 0)
	{
		c = a % b;
		a = b;
		b = c;
	}
	return a;
}

__device__ __host__ int lcm(int a, int b)
{
    return a * b / gcd(a, b);
}




CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}



__device__ void ViewMaskData(ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, int tid, int nPrarticlePosCntCuda, int nXFileVoxCnt, int nYFileVoxCnt)
{

	float f[7] = {pstPrarticlePosCudaMask[tid].fPorosity,0,0,0,0,0,0};
	float fW[7] = {pstPrarticlePosCudaMask[tid].fHaveWater,0,0,0,0,0,0};

	int nValue = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[1] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[1] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[2] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[2] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[3] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[3] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[4] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[4] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[5] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[5] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[6] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[6] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}


	printf( "New Tid : %d -> Porosity:%f,%f,%f,%f,%f,%f,%f\nWater:%f,%f,%f,%f,%f,%f,%f\n", tid, 
		f[0],
		f[1],
		f[2],
		f[3],
		f[4],
		f[5],
		f[6],
		fW[0],
		fW[1],
		fW[2],
		fW[3],
		fW[4],
		fW[5],
		fW[6]
		);
    
}



__device__ void InputMaskDataToSharedMemValue(ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, 
											  ST_PARTICLE_POS_UNIT_PROCESS	*astParticle_pos_unitProcess, 
											  int nIdx,
											  int threadIdxT, int nPrarticlePosCntCuda)
{
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCudaMask[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdxT].fPorosity;
			pstPrarticlePosCudaMask[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdxT].fHaveWater;
		}
		
	}
}



__global__ void kernelCalcRocking(int nStep, 
								  int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, 
								  int nXFileVoxCnt, int nYFileVoxCnt, int nZFileVoxCnt,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	//return;

	 // ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int tid = blockIdx.x * blockDim.x + threadIdx.x + nStep;

	//if(tid > nPrarticlePosCntCuda)
	//	return;

	
	//return;
	//int nExternalSideIdx = threadIdx.x % 6;
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid >= nPrarticlePosCntCuda)
		return;

	

	pstPrarticlePosCuda[tid].bCheck = true;


	if(pstPrarticlePosCuda[tid].sStoneType == -1)
		return;
	
	//printf("BlockIdx: %d\t BlcokDim: %d\t ThreadIdx: %d\t Step: %d\tMax(%d)\ttid : %d\n", blockIdx.x, blockDim.x, threadIdx.x, nStep, nPrarticlePosCntCuda, tid);
	
	int nThreaIdxSharedMem = threadIdx.x * 7;

	//extern __shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	//__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[32 * 7]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[dfSHARED_MEMORY_SIZE]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * dfSHARED_MEMORY_SIZE);
	//memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * lcm(nThreadCnt, 6));
	__syncthreads();

	

	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
	//if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
	if(pstPrarticlePosCuda[tid].abExternalSide[0] ||
		pstPrarticlePosCuda[tid].abExternalSide[1] ||
		pstPrarticlePosCuda[tid].abExternalSide[2] ||
		pstPrarticlePosCuda[tid].abExternalSide[3] ||
		pstPrarticlePosCuda[tid].abExternalSide[4] ||
		pstPrarticlePosCuda[tid].abExternalSide[5] ) //! �ܺδ� ���� �Ի�ر�
	{
		//! �ܺ� �Ի�ر��� �������� ó���Ѵ�.(SHARED ����ŷ ���� �޸𸮿� ó��!)
		 astParticle_pos_unitProcess[nThreaIdxSharedMem].fPorosity = (pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);
		
		 pstPrarticlePosCuda[tid].abEffectPorosity[0] = pstPrarticlePosCuda[tid].abExternalSide[0] || pstPrarticlePosCuda[tid].abExternalSide[1] || pstPrarticlePosCuda[tid].abExternalSide[2] || pstPrarticlePosCuda[tid].abExternalSide[3] || pstPrarticlePosCuda[tid].abExternalSide[4] || pstPrarticlePosCuda[tid].abExternalSide[5];

		 
	//printf("Side: %d\t%d\t%d\t%d\t%d\t%d\tTid:%d\n",
	//	pstPrarticlePosCuda[tid].abExternalSide[0],
	//	pstPrarticlePosCuda[tid].abExternalSide[1],
	//	pstPrarticlePosCuda[tid].abExternalSide[2],
	//	pstPrarticlePosCuda[tid].abExternalSide[3],
	//	pstPrarticlePosCuda[tid].abExternalSide[4],
	//	pstPrarticlePosCuda[tid].abExternalSide[5],
	//	tid
	//	);

		
		 //printf("BlockIdx: %d\t BlcokDim: %d\t ThreadIdx: %d\t Step: %d\tMax(%d)\ttid : %d\n", blockIdx.x, blockDim.x, nThreaIdxSharedMem, nStep, nPrarticlePosCntCuda, tid);

		//printf( "OuterMask : %d -> Porosity:%f\n", tid, astParticle_pos_unitProcess[nThreaIdxSharedMem].fPorosity);
	}
	else //! ���� ó��
	{
		//return;

		if(pstPrarticlePosCuda[tid].sStoneType == 0 ) //! ���ظ� ó���ؾ��Ѵ�. //! 1.0�� ������ ������ ���� ������!
		{
			pstPrarticlePosCuda[tid].fPorosity = 0;

			//printf( "InnerMask Check : %d(%f)\n", tid, astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater);


			//__shared__ unsigned int sdata[6];

			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
		
			//printf("Water Temp : %f\tLayer : %d\n", fHaveWaterTemp, pstPrarticlePosCuda[tid].sLayerIdx);

			//! 0���� ������ ������ ���� ������
			if(fHaveWaterTemp < 0.0)
				fHaveWaterTemp = 0.0;

			//! ����ŷ�� ����
			astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater += fHaveWaterTemp;

			


			//fPorosity = 0.0;
			//__syncthreads();


			//printf( "Inner : %d -> Water:%f\n", tid, astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater);

			//(pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);


			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
			{

				//! ������ ó���Ǹ� ���� �Լ����� �������� �������ֱ�����!
				//astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater = -0.5;

				//[20190719] kjky12 �ϴ� ���� ó���Ǵ°ɷ� ó���Ǹ� �����Լ����� ������ �����Ѵ�. -> �׷��� �ٽ� -1�� ����������ҵ�	
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fHaveWater = 1.0;
				astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fPorosity += (fCalcWaterChange / 5.0);

				pstPrarticlePosCuda[tid].abEffectPorosity[1] = true;
			}
			
			//int nChangeCnt = 1;
			//int nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fPorosity >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}
			//}
			//

			//nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}
			//}
			//

			//nIdx = tid - nXFileVoxCnt;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}

			//}


			//nIdx = tid + nXFileVoxCnt;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}
			//}
			//

			//nIdx = tid - 1;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}
			//}
			//
			//nIdx = tid + 1;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			nChangeCnt += 1;
			//	}
			//}

			////////////////////////////
			////! ī���� �ȰŸ� ���� �Լ����� �Ի�ر��� �Ͼ ���������ؼ� ������ �־���
			//nIdx = tid;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}

			//nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}
			//

			//nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}
			//

			//nIdx = tid - nXFileVoxCnt;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}

			//}


			//nIdx = tid + nXFileVoxCnt;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}
			//

			//nIdx = tid - 1;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}
			//
			//nIdx = tid + 1;
			//if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			//{
			//	if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
			//	{
			//		if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
			//			astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / nChangeCnt;
			//	}
			//}

			//printf("tid:%d(%d)->%f\n", tid, nChangeCnt, pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1));
		}

	}

/*	ViewMaskData(pstPrarticlePosCudaMask, tid, nPrarticlePosCntCuda, nXFileVoxCnt, nYFileVoxCnt);
	return;*/
	

	//printf("HaveWater: %f\t%f\t%f\t%f\t%f\t%f\t%f\tTid:%d(%d)\n",
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+1].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+2].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+3].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+4].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+5].fHaveWater,
	//	astParticle_pos_unitProcess[nThreaIdxSharedMem+6].fHaveWater,
	//	tid, pstPrarticlePosCuda[tid].sStoneType
	//	);

	

	int nIdx = tid;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem].fPorosity;
			//! ���� �����Լ����� ���� �־������
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem].fHaveWater;

			//printf( "InnerMask Check : %d(%f)\n", tid, pstPrarticlePosCuda[nIdx].fHaveWater);
		}
	}

	nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 1].fHaveWater;
		}
	}
	

	nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 2].fHaveWater;
		}
	}
	

	nIdx = tid - nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 3].fHaveWater;
		}

	}


	nIdx = tid + nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 4].fHaveWater;
		}
	}
	

	nIdx = tid - 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 5].fHaveWater;
		}
	}
	
	nIdx = tid + 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[nThreaIdxSharedMem + 6].fHaveWater;
		}
	}


	//ViewMaskData(pstPrarticlePosCudaMask, tid, nPrarticlePosCntCuda, nXFileVoxCnt, nYFileVoxCnt);

} 


__global__ void kernelCalcRockingShared(int nStep, 
								  int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, 
								  int nXFileVoxCnt, int nYFileVoxCnt, int nZFileVoxCnt,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	

} 


__global__ void kernelCalcRockingMasking(int nStep, 
										 int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int tid = blockIdx.x * blockDim.x + threadIdx.x + nStep;

	if(tid >= nPrarticlePosCntCuda)
		return;

	//pstPrarticlePosCuda[tid].bCheck = true;


	if(pstPrarticlePosCuda[tid].sStoneType == -1)
		return;


	//printf("Prev:%03d->%f\n",tid, pstPrarticlePosCuda[tid].fPorosity);

	//pstPrarticlePosCuda[tid].fPorosity = pstPrarticlePosCuda[tid].fPorosity + pstPrarticlePosCudaMask[tid].fPorosity;
	//pstPrarticlePosCuda[tid].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater + pstPrarticlePosCudaMask[tid].fHaveWater;
	
	//if( pstPrarticlePosCuda[tid].sStoneType == 0
	//	||(pstPrarticlePosCuda[tid].abExternalSide[0] || pstPrarticlePosCuda[tid].abExternalSide[1] || pstPrarticlePosCuda[tid].abExternalSide[2] || pstPrarticlePosCuda[tid].abExternalSide[3] || pstPrarticlePosCuda[tid].abExternalSide[4] || pstPrarticlePosCuda[tid].abExternalSide[5])
	//) //! �����ε� �ܺο� ��� ��� �ܺη� �Ǵ�
	//{
	//	pstPrarticlePosCuda[tid].sStoneType = -1;
	//}


	//if(pstPrarticlePosCuda[tid].abExternalSide[0] && pstPrarticlePosCuda[tid].abExternalSide[1] && pstPrarticlePosCuda[tid].abExternalSide[2]
	//&& pstPrarticlePosCuda[tid].abExternalSide[3] && pstPrarticlePosCuda[tid].abExternalSide[4] && pstPrarticlePosCuda[tid].abExternalSide[5]) //! 6���� �ܺ��ΰ�� �׳� �ܺη� �Ǵ�
	//{
	//	pstPrarticlePosCuda[tid].sStoneType = -1;
	//}

	if(pstPrarticlePosCuda[tid].sStoneType != 0 )
	{
		//pstPrarticlePosCuda[tid].fPorosity += pstPrarticlePosCudaMask[tid].fPorosity;

		if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration ) // �Ի�ر� ���ް��� �����Ͽ� ����
		{
			printf("%d/%d StonType:%d\tPrositiy : %f\tWater : %f[tid]\n", pstPrarticlePosCuda[tid].abEffectPorosity[0], pstPrarticlePosCuda[tid].abEffectPorosity[1],
			pstPrarticlePosCuda[tid].sStoneType
			, pstPrarticlePosCuda[tid].fPorosity, pstPrarticlePosCuda[tid].fHaveWater, tid);

			
			//if(pstPrarticlePosCudaMask[tid].fHaveWater > 0.0)
			//{
			//	//pstPrarticlePosCuda[tid].sStoneType = -2;
			//	pstPrarticlePosCuda[tid].sStoneType = 0; //! �Ѱ��� ������ ����
			//	//! �Ի�ر��� �Ͼ ������ �� ������ �����Լ����� 0.0���� �ʱ�ȭ
			//	pstPrarticlePosCuda[tid].fHaveWater = 0.0;

			//	pstPrarticlePosCuda[tid].fHaveWater = pstPrarticlePosCudaMask[tid].fHaveWater;
			//}
			//else
			//{
			//	pstPrarticlePosCuda[tid].sStoneType = -1;
			//}


			//printf("%3d\t%3d\t%3d\t%d%d%d%d%d%d\t%d\tTid-%d(%f)\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,
			//	pstPrarticlePosCuda[tid].abExternalSide[0],
			//	pstPrarticlePosCuda[tid].abExternalSide[1],
			//	pstPrarticlePosCuda[tid].abExternalSide[2],
			//	pstPrarticlePosCuda[tid].abExternalSide[3],
			//	pstPrarticlePosCuda[tid].abExternalSide[4],
			//	pstPrarticlePosCuda[tid].abExternalSide[5],
			//	pstPrarticlePosCuda[tid].sStoneType,
			//	tid,
			//	pstPrarticlePosCuda[tid].fHaveWater
			//	);

			if(pstPrarticlePosCuda[tid].abExternalSide[0] ||
				pstPrarticlePosCuda[tid].abExternalSide[1] ||
				pstPrarticlePosCuda[tid].abExternalSide[2] ||
				pstPrarticlePosCuda[tid].abExternalSide[3] ||
				pstPrarticlePosCuda[tid].abExternalSide[4] ||
				pstPrarticlePosCuda[tid].abExternalSide[5] ) //! ���� �ܺθ� �ܺ� �Ի�ر�
			{
				pstPrarticlePosCuda[tid].sStoneType = -1;
			}
			else
			{
				//printf("TID\t%d\t->%f\n", tid, pstPrarticlePosCudaMask[tid].fHaveWater);


				pstPrarticlePosCuda[tid].sStoneType = 0; //! �Ѱ��� ������ ����
				//! �Ի�ر��� �Ͼ ������ �� ������ �����Լ����� 0.0���� �ʱ�ȭ
				pstPrarticlePosCuda[tid].fHaveWater = 0.0;

				//pstPrarticlePosCuda[tid].fHaveWater = pstPrarticlePosCudaMask[tid].fHaveWater;
				
				
			}
				

				//if(pstPrarticlePosCuda[tid].sStoneType == -2) //-2 �Ի�ر��� �Ͼ �ű� ������ -> ���� �Ǵ� �ܺ� ���� ���ڷ� ����������Ѵ�
				//{
				//	if(pstPrarticlePosCuda[tid].abExternalSide[0] ||
				//		pstPrarticlePosCuda[tid].abExternalSide[1] || 
				//		pstPrarticlePosCuda[tid].abExternalSide[2] || 
				//		pstPrarticlePosCuda[tid].abExternalSide[3] || 
				//		pstPrarticlePosCuda[tid].abExternalSide[4] || 
				//		pstPrarticlePosCuda[tid].abExternalSide[5])
				//	{
				//		 //! �Ѱ��� ������ �ܺ� ����!
				//		pstPrarticlePosCuda[tid].sStoneType = -1;
				//	}
				//	else
				//	{
				//		
				//		pstPrarticlePosCuda[tid].sStoneType = 0; //! �Ѱ��� ������ ����
				//		//! �Ի�ر��� �Ͼ ������ �� ������ �����Լ����� 0.0���� �ʱ�ȭ
				//		pstPrarticlePosCuda[tid].fHaveWater = 0.0;
				//	}

				//	printf("%3d\t%3d\t%3d\t%d%d%d%d%d%d\t%d\tTid-%d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,
				//	pstPrarticlePosCuda[tid].abExternalSide[0],
				//	pstPrarticlePosCuda[tid].abExternalSide[1],
				//	pstPrarticlePosCuda[tid].abExternalSide[2],
				//	pstPrarticlePosCuda[tid].abExternalSide[3],
				//	pstPrarticlePosCuda[tid].abExternalSide[4],
				//	pstPrarticlePosCuda[tid].abExternalSide[5],
				//	pstPrarticlePosCuda[tid].sStoneType,
				//	tid);

				//}
				//else
				//{
				//	//printf("%3d\t%3d\t%3d\t%d%d%d%d%d%d\t%d\tNo-%d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,
				//	//pstPrarticlePosCuda[tid].abExternalSide[0],
				//	//pstPrarticlePosCuda[tid].abExternalSide[1],
				//	//pstPrarticlePosCuda[tid].abExternalSide[2],
				//	//pstPrarticlePosCuda[tid].abExternalSide[3],
				//	//pstPrarticlePosCuda[tid].abExternalSide[4],
				//	//pstPrarticlePosCuda[tid].abExternalSide[5],
				//	//pstPrarticlePosCuda[tid].sStoneType,
				//	//tid);
				//}

			
		}
	}


	

	//printf("Afte:%03d->%f\n",tid, pstPrarticlePosCuda[tid].fPorosity);

}

__global__ void kernelReCalcExternalSide(int nStep, 
										 int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda
										 ,int nXFileVoxCnt, int nYFileVoxCnt)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int tid = blockIdx.x * blockDim.x + threadIdx.x + nStep;

	if(tid >= nPrarticlePosCntCuda)
		return;

	//pstPrarticlePosCuda[tid].bCheck = true;


	//pstPrarticlePosCuda[tid].abExternalSide[0];
	//memset(pstPrarticlePosCuda[tid].abExternalSide, false, sizeof(bool) * 6);
	

	//printf("tid : %d\n", tid);
		
	//! �����¿�յ��� ������ �ִ��� Ȯ���ϰ� �ܺ� ���� �ܸ��� üũ�Ѵ�. 

	int nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN2\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[0] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[0] = false;
		}
	}
	
	
	
	nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
	//printf("Idx : %d\ttid : %d\n", nIdx, tid);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN1\n\n\n");
		
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[1] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[1] = false;
		}
	}

	
	
	nIdx = tid - nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN3\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[2] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[2] = false;
		}
	}

	nIdx = tid + nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN4\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[3] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[3] = false;
		}
	}

	nIdx = tid - 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[4] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[4] = false;
		}
	}
	
	nIdx = tid + 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[5] = true;
		}
		else
		{
			pstPrarticlePosCuda[tid].abExternalSide[5] = false;
		}
	}

	

	//if(pstPrarticlePosCuda[tid].sStoneType == 0 &&
	//	(pstPrarticlePosCuda[tid].abExternalSide[0] || pstPrarticlePosCuda[tid].abExternalSide[1] ||
	//			pstPrarticlePosCuda[tid].abExternalSide[2] || pstPrarticlePosCuda[tid].abExternalSide[3] ||
	//			pstPrarticlePosCuda[tid].abExternalSide[4] || pstPrarticlePosCuda[tid].abExternalSide[5] )) //! �����ε� �ܺο� ��������� �ܺη� ����
	//		{
	//			pstPrarticlePosCuda[tid].sStoneType = -1;
	//	}


}


void CGPUCalcRockAgingInner::SetInnderVoxelData(int nRepeatCnt, int nPrarticlePosCnt, ST_PARTICLE_POS_CUDA	*pstPrarticlePos, ST_PARTICLE_POS_CUDA	*pstPrarticlePosMask)
{
	//! ���� ����
	ST_PARTICLE_POS_CUDA *pstPrarticlePosCuda;
	ST_PARTICLE_POS_CUDA *pstPrarticlePosCudaMask;
	//! ���� ����
	//int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS_CUDA);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}
	


	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}


	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Memset \n" );
	}


	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}*/
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	if ( hipSuccess != hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice))
	{
		printf( "Error! Copy1 \n" );
	}

	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);

	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}
	
	//! �ּ� �����
	int nThreadCnt = 32; // 32�� ���� ó��
	//int nSharedMemoryCnt = lcm(nThreadCnt, 6); //92
	int nBlockCnt = (nPrarticlePosCnt / nThreadCnt) + 1;
	//kernelCalcRocking<<<nBlockCnt, nThreadCnt, nThreadCnt * 7>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	//int nBlockStep = 256;
	int nBlockStep = 128;
	
	for(int a = 0; a< nRepeatCnt; a++)
	{
		int nStep = 0;

		/// �ΰ��� ������ ó���� �Ǿ���Ѵ�.
		//------------------------------------------------------------------------
		//! ���� �޸𸮸� ���� ��� ó���Ǵ� ��ƾ
		nStep = 0;
		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			kernelCalcRocking<<<nBlockStepTemp, nThreadCnt>>>(nStep, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, 
				m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, 
				m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, 
				m_fCalcWaterInnerAbsorption, 
				m_fCalcLayerWaterAborption, m_fCalcWaterChange);	

			nStep += nBlockStepTemp * nThreadCnt;

			if(n % dfCALC_BLOCK_CNT == dfCALC_BLOCK_CNT - 1)
			{
				hipDeviceSynchronize();
			}
		}
		//------------------------------------------------------------------------

		//------------------------------------------------------------------------
		//! ���� �޸𸮸� �Ϻζ� �����Ǵ� ��ƾ ��������
		//------------------------------------------------------------------------

		

		//int nBlockStepTemp = nBlockStep;	
		//kernelCalcRocking<<<nBlockStepTemp, nThreadCnt>>>(nStep, nPrarticlePosCnt, pstPrarticlePosCuda/*, pstPrarticlePosCudaMask*/, 
		//	m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, 
		//	m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, 
		//	m_fCalcWaterInnerAbsorption, 
		//	m_fCalcLayerWaterAborption, m_fCalcWaterChange);	
		//
		//
		//nStep += nBlockStepTemp * nThreadCnt;


		//nBlockStepTemp = nBlockStep;	
		//kernelCalcRocking<<<nBlockStepTemp, nThreadCnt>>>(nStep, nPrarticlePosCnt, pstPrarticlePosCuda/*, pstPrarticlePosCudaMask*/, 
		//	m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, 
		//	m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, 
		//	m_fCalcWaterInnerAbsorption, 
		//	m_fCalcLayerWaterAborption, m_fCalcWaterChange);	

		
		
		nStep = 0;
		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			
			kernelCalcRockingMasking<<<nBlockStepTemp, nThreadCnt>>>(nStep, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask);
			nStep += nBlockStepTemp * nThreadCnt;

			if(n % dfCALC_BLOCK_CNT == dfCALC_BLOCK_CNT - 1)
			{
				hipDeviceSynchronize();
			}

		}

		nStep = 0;
		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			
			kernelReCalcExternalSide<<<nBlockStepTemp, nThreadCnt>>>(nStep, nPrarticlePosCnt, pstPrarticlePosCuda, m_nXFileVoxCnt, m_nYFileVoxCnt);
			nStep += nBlockStepTemp * nThreadCnt;

			if(n % dfCALC_BLOCK_CNT == dfCALC_BLOCK_CNT - 1)
			{
				hipDeviceSynchronize();
			}
		}

		printf("\n\n");

		//Sleep(100);
	}
		

	

	//hipDeviceSynchronize();

	//hipDeviceSynchronize();
	//hipStreamSynchronize(cudaStream);


	//for(int n = 0; n < nPrarticlePosCnt; n++)
	//{
	//	printf("Acess idx : %d -> Porosity:%f,%f\n", n, pstPrarticlePosCudaMask[n].fPorosity, pstPrarticlePosCudaMask[n].fHaveWater);
	//}
	//printf("TEST\n");
	
	//memset(pstPrarticlePos, NULL, nSizeCnt*nPrarticlePosCnt);

	if ( hipSuccess != hipMemcpy(pstPrarticlePos, pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost))
	{
		printf( "Error! Copy2 \n" );
	}

	//hipDeviceSynchronize();

	hipFree(pstPrarticlePosCuda);
	//hipFree(pstPrarticlePosCudaMask);
	//hipFree(pnPrarticlePosCntCuda);

	//pstPrarticlePosMask[0].x = 1;



}


