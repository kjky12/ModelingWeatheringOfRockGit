#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"

#define dfSHARED_MEMORY_SIZE	32*7

__device__ __host__ int gcd(int a, int b)
{
	int c;
	while (b != 0)
	{
		c = a % b;
		a = b;
		b = c;
	}
	return a;
}

__device__ __host__ int lcm(int a, int b)
{
    return a * b / gcd(a, b);
}




CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}


//__global__ void kernelCalcRocking(int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
//								  int nX, int nY, int nZ,
//								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
//								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
//								  )
//{ 
//	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
//	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	int nExternalSideIdx = threadIdx.x;
//    //printf( "x:%d\n", nPrarticlePosCntCuda);
//
//	if(tid > nPrarticlePosCntCuda)
//		return;
//
//	
//    
//
//
//	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
//	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
//	{
//		__shared__ float			fPorosity;
//		fPorosity = 0.0;
//
//
//		if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == TRUE)
//		{
//			//switch(stParticlePos.vecExternalSide[nEx])	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			{
//			case 0:
//				{
//					fPorosity += (fCoefficient * fTopRate);
//				}
//				break;
//			case 1:
//				{
//					fPorosity += (fCoefficient * fBottomRate);
//				}
//				break;
//			case 2:
//			case 3:
//			case 4:
//			case 5:
//				{
//					fPorosity += (fCoefficient * fSideRate);
//				}
//				break;
//			default:
//				break;
//			}
//
//		}
//
//        __syncthreads();
//
//		pstPrarticlePosCudaMask[tid].fPorosity = fPorosity;
//		printf( "OuterMask : %d\n", tid);
//
//		//fPorosity ���ط� >= fGranularDisintegration �Ի�ر� ���ް�
//		//if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//		//{
//		//	
//
//		//	//stParticlePos.bUse = false;
//
//		//	//strKey.Format(L"%d-%d-%d",x,y,z);
//
//		//	//vecDeleParticle.push_back(strKey);
//
//		//	//break;
//		//}
//	}
//	else //! ���� ó��
//	{
//		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
//		{
//			printf( "InnerMask Check : %d\n", tid);
//
//
//			//__shared__ unsigned int sdata[6];
//
//			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
//			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
//			
//			//! 0���� ������ ������ ���� ������
//			if(fHaveWaterTemp < 0.0)
//				fHaveWaterTemp = 0.0;
//
//			__shared__ float			fPorosity[6];
//			memset(fPorosity, NULL, sizeof(float) * 6);
//			//fPorosity = 0.0;
//			//__syncthreads();
//
//			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
//			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
//			{
//				int nPos = tid;
//				switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//				{
//				case 0:
//					nPos += (nX * nY);
//					break;
//				case 1:
//					nPos -= (nX * nY);
//					break;
//				case 2:
//					nPos += nX;
//					break;
//				case 3:
//					nPos -= nX;
//					break;
//				case 4:
//					nPos -= 1;
//					break;
//				case 5:
//					nPos += 1;
//					break;
//				default:
//					break;
//				}
//
//				if(nPos > nPrarticlePosCntCuda)
//					return;
//
//				fPorosity[nExternalSideIdx] = (fCalcWaterChange / 5.0);
//
//				__syncthreads();
//
//				pstPrarticlePosCudaMask[nPos].fPorosity += fPorosity[nExternalSideIdx];
//
//				//if(pstPrarticlePosCudaMask[nPos].fPorosity >= pstPrarticlePosCudaMask[nPos].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//				//{
//				//	nBreakCnt++;
//
//				//	__syncthreads();
//				//	pstPrarticlePosCudaMask[nPos].fHaveWater = (pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) / (float)(nBreakCnt + 1);
//				//	printf( "InnerMask : %d\n", nPos);
//				//}
//			
//
//				//pstPrarticlePosCudaMask[nPos]
//
//			}
//
//		}
//	}
//
//	/*if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == true)
//	{
//
//	}*/
//
//
//	//pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx];
//
//    //printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);
//
//
//} 
//
//
//
//void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
//{
//	//! ���� ����
//	ST_PARTICLE_POS *pstPrarticlePosCuda;
//	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
//	//! ���� ����
//	//int *pnPrarticlePosCntCuda;
//
//	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
//	int nSizeCnt = sizeof(ST_PARTICLE_POS);
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//
//	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Memset \n" );
//	}
//
//	
//
//
//	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
//	{
//		printf( "Error! Malloc \n" );
//
//		Sleep(1000);
//	}*/
//	//else
//	//{
//	//	printf( "ErrorGOOD \n" );
//
//	//}
//
//
//	
//	
//	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
//	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
//	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);
//
//
//
//	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
//	//{
//	//	printf( "Error! Y VoxelState Memcpy \n" );
//	//}
//
//	
//	int nBlockCnt = nPrarticlePosCnt;
//	kernelCalcRocking<<<nBlockCnt, 6>>>(nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
//	
//
//	
//	hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost);
//
//
//	hipFree(pstPrarticlePosCuda);
//	hipFree(pstPrarticlePosCudaMask);
//	//hipFree(pnPrarticlePosCntCuda);
//
//	int a= 0;
//
//
//}


__device__ void ViewMaskData(ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, int tid, int nPrarticlePosCntCuda, int nXFileVoxCnt, int nYFileVoxCnt)
{

	float f[7] = {pstPrarticlePosCudaMask[tid].fPorosity,0,0,0,0,0,0};
	float fW[7] = {pstPrarticlePosCudaMask[tid].fHaveWater,0,0,0,0,0,0};

	int nValue = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[1] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[1] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[2] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[2] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[3] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[3] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[4] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[4] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[5] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[5] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[6] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[6] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}


	printf( "New Tid : %d -> Porosity:%f,%f,%f,%f,%f,%f,%f\nWater:%f,%f,%f,%f,%f,%f,%f\n", tid, 
		f[0],
		f[1],
		f[2],
		f[3],
		f[4],
		f[5],
		f[6],
		fW[0],
		fW[1],
		fW[2],
		fW[3],
		fW[4],
		fW[5],
		fW[6]
		);
    
}

__device__ void InputMaskDataToSharedMem(ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, ST_PARTICLE_POS_UNIT_PROCESS	*astParticle_pos_unitProcess, int tid, int threadIdx, int nPrarticlePosCntCuda, int nXFileVoxCnt, int nYFileVoxCnt)
{

	pstPrarticlePosCudaMask[tid].fPorosity += astParticle_pos_unitProcess[threadIdx].fPorosity;
	pstPrarticlePosCudaMask[tid].fHaveWater += astParticle_pos_unitProcess[threadIdx].fHaveWater;


	int nValue = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 1].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 1].fHaveWater;

	

	}

	nValue = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{

		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 2].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 2].fHaveWater;

	
	}

	nValue = tid + nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 3].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 3].fHaveWater;

		
	}

	nValue = tid - nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 4].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 4].fHaveWater;

	}

	nValue = tid - 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 5].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 5].fHaveWater;


	}

	nValue = tid + 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 6].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 6].fHaveWater;
	}

	
	

}


__device__ void InputMaskDataToSharedMemValue(ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, 
											  ST_PARTICLE_POS_UNIT_PROCESS	*astParticle_pos_unitProcess, 
											  int nIdx,
											  int threadIdxT, int nPrarticlePosCntCuda)
{
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCudaMask[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdxT].fPorosity;
			pstPrarticlePosCudaMask[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdxT].fHaveWater;
		}
		
	}
}



__global__ void kernelCalcRocking(int nThreadCnt, 
								  int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask, 
								  int nXFileVoxCnt, int nYFileVoxCnt, int nZFileVoxCnt,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	//return;

	 // ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//if(tid > nPrarticlePosCntCuda)
	//	return;

	
	//return;
	//int nExternalSideIdx = threadIdx.x % 6;
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid >= nPrarticlePosCntCuda)
		return;

	if(pstPrarticlePosCuda[tid].sStoneType == -1)
		return;

	//printf("tid : %d\n", tid);

	//extern __shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	//__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[32 * 7]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[dfSHARED_MEMORY_SIZE]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * dfSHARED_MEMORY_SIZE);
	//memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * lcm(nThreadCnt, 6));
	__syncthreads();

	

	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
	{
		//! �ܺ� �Ի�ر��� �������� ó���Ѵ�.(SHARED ����ŷ ���� �޸𸮿� ó��!)
		 astParticle_pos_unitProcess[threadIdx.x].fPorosity = (pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);
		
		
		//printf( "OuterMask : %d -> Porosity:%f\n", tid, astParticle_pos_unitProcess[threadIdx.x].fPorosity);
	}
	else //! ���� ó��
	{
		return;

		if(pstPrarticlePosCuda[tid].sStoneType == 0 ) //! ���ظ� ó���ؾ��Ѵ�. //! 1.0�� ������ ������ ���� ������!
		{
			//printf( "InnerMask Check : %d\n", tid);


			//__shared__ unsigned int sdata[6];

			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
		
			//printf("Water Temp : %f\tLayer : %d\n", fHaveWaterTemp, pstPrarticlePosCuda[tid].sLayerIdx);

			//! 0���� ������ ������ ���� ������
			if(fHaveWaterTemp < 0.0)
				fHaveWaterTemp = 0.0;

			//! ����ŷ�� ����
			astParticle_pos_unitProcess[threadIdx.x].fHaveWater += fHaveWaterTemp;



			//fPorosity = 0.0;
			//__syncthreads();


			//printf( "Inner : %d -> Water:%f\n", tid, astParticle_pos_unitProcess[threadIdx.x].fHaveWater);

			//(pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);


			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
			{
				//! ������ ó���Ǹ� ���� �Լ����� �������� �������ֱ�����!
				//astParticle_pos_unitProcess[threadIdx.x].fHaveWater = -0.5;

				//[20190719] kjky12 �ϴ� ���� ó���Ǵ°ɷ� ó���Ǹ� �����Լ����� ������ �����Ѵ�. -> �׷��� �ٽ� -1�� ����������ҵ�	
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 1].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 2].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 3].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 4].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 5].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				//astParticle_pos_unitProcess[threadIdx.x + 6].fHaveWater += 1.0;
				astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity += (fCalcWaterChange / 5.0);
			}
			
			int nChangeCnt = 0;
			int nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}
			}
			

			nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}
			}
			

			nIdx = tid - nXFileVoxCnt;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}

			}


			nIdx = tid + nXFileVoxCnt;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}
			}
			

			nIdx = tid - 1;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}
			}
			
			nIdx = tid + 1;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						nChangeCnt += 1;
				}
			}

			//////////////////////////
			//! ī���� �ȰŸ� ���� �Լ����� �Ի�ر��� �Ͼ ���������ؼ� ������ �־���
			nIdx = tid;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					astParticle_pos_unitProcess[threadIdx.x].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}

			nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 1].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}
			

			nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 2].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}
			

			nIdx = tid - nXFileVoxCnt;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 3].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}

			}


			nIdx = tid + nXFileVoxCnt;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 4].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}
			

			nIdx = tid - 1;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 5].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}
			
			nIdx = tid + 1;
			if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
			{
				if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
				{
					if(pstPrarticlePosCuda[nIdx].fPorosity + astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity  >= pstPrarticlePosCuda[nIdx].fGranularDisintegration)
						astParticle_pos_unitProcess[threadIdx.x + 6].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1);
				}
			}

			//printf("tid:%d(%d)->%f\n", tid, nChangeCnt, pstPrarticlePosCuda[tid].fHaveWater / (nChangeCnt + 1));
		}

	}

/*	ViewMaskData(pstPrarticlePosCudaMask, tid, nPrarticlePosCntCuda, nXFileVoxCnt, nYFileVoxCnt);
	return;*/
	

	int nIdx = tid;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x].fPorosity;
			//! ���� �����Լ����� ���� �־������
			pstPrarticlePosCuda[nIdx].fHaveWater = astParticle_pos_unitProcess[threadIdx.x].fHaveWater;
		}
	}

	nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 1].fHaveWater;
		}
	}
	

	nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 2].fHaveWater;
		}
	}
	

	nIdx = tid - nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 3].fHaveWater;
		}

	}


	nIdx = tid + nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 4].fHaveWater;
		}
	}
	

	nIdx = tid - 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 5].fHaveWater;
		}
	}
	
	nIdx = tid + 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType != -1)
		{
			pstPrarticlePosCuda[nIdx].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity;
			pstPrarticlePosCuda[nIdx].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 6].fHaveWater;
		}
	}


	//ViewMaskData(pstPrarticlePosCudaMask, tid, nPrarticlePosCntCuda, nXFileVoxCnt, nYFileVoxCnt);

} 


__global__ void kernelCalcRockingMasking(int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCudaMask)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid >= nPrarticlePosCntCuda)
		return;

	if(pstPrarticlePosCuda[tid].sStoneType == -1)
		return;


	//printf("Prev:%03d->%f\n",tid, pstPrarticlePosCuda[tid].fPorosity);

	//pstPrarticlePosCuda[tid].fPorosity = pstPrarticlePosCuda[tid].fPorosity + pstPrarticlePosCudaMask[tid].fPorosity;
	//pstPrarticlePosCuda[tid].fHaveWater = pstPrarticlePosCuda[tid].fHaveWater + pstPrarticlePosCudaMask[tid].fHaveWater;
	
	//if( pstPrarticlePosCuda[tid].sStoneType == 0
	//	||(pstPrarticlePosCuda[tid].abExternalSide[0] || pstPrarticlePosCuda[tid].abExternalSide[1] || pstPrarticlePosCuda[tid].abExternalSide[2] || pstPrarticlePosCuda[tid].abExternalSide[3] || pstPrarticlePosCuda[tid].abExternalSide[4] || pstPrarticlePosCuda[tid].abExternalSide[5])
	//) //! �����ε� �ܺο� ��� ��� �ܺη� �Ǵ�
	//{
	//	pstPrarticlePosCuda[tid].sStoneType = -1;
	//}


	//if(pstPrarticlePosCuda[tid].abExternalSide[0] && pstPrarticlePosCuda[tid].abExternalSide[1] && pstPrarticlePosCuda[tid].abExternalSide[2]
	//&& pstPrarticlePosCuda[tid].abExternalSide[3] && pstPrarticlePosCuda[tid].abExternalSide[4] && pstPrarticlePosCuda[tid].abExternalSide[5]) //! 6���� �ܺ��ΰ�� �׳� �ܺη� �Ǵ�
	//{
	//	pstPrarticlePosCuda[tid].sStoneType = -1;
	//}

	if(/*pstPrarticlePosCuda[tid].sStoneType != 0 &&*/ pstPrarticlePosCuda[tid].sStoneType != -1 )
	{
		if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration ) // �Ի�ر� ���ް��� �����Ͽ� ����
		{
			pstPrarticlePosCuda[tid].sStoneType = -2;
		}
	}


	

	//printf("Afte:%03d->%f\n",tid, pstPrarticlePosCuda[tid].fPorosity);

}

__global__ void kernelReCalcExternalSide(int nPrarticlePosCntCuda, ST_PARTICLE_POS_CUDA	*pstPrarticlePosCuda
										 ,int nXFileVoxCnt, int nYFileVoxCnt)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid >= nPrarticlePosCntCuda)
		return;
	//pstPrarticlePosCuda[tid].abExternalSide[0];
	memset(pstPrarticlePosCuda[tid].abExternalSide, false, sizeof(bool) * 6);
	

	//printf("tid : %d\n", tid);
		
	//! �����¿�յ��� ������ �ִ��� Ȯ���ϰ� �ܺ� ���� �ܸ��� üũ�Ѵ�. 

	int nIdx = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN2\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[0] = true;
		}
	}
	
	
	
	nIdx = tid - (nXFileVoxCnt * nYFileVoxCnt);
	//printf("Idx : %d\ttid : %d\n", nIdx, tid);
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN1\n\n\n");
		
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[1] = true;
		}
	}

	
	
	nIdx = tid - nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN3\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[2] = true;
		}
	}

	nIdx = tid + nXFileVoxCnt;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		//printf("INNNNN4\n\n\n");
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[3] = true;
		}
	}

	nIdx = tid - 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[4] = true;
		}
	}
	
	nIdx = tid + 1;
	if(nIdx <= nPrarticlePosCntCuda && nIdx >= 0)
	{
		if(pstPrarticlePosCuda[nIdx].sStoneType == -1)
		{
			pstPrarticlePosCuda[tid].abExternalSide[5] = true;
		}
	}

	if(pstPrarticlePosCuda[tid].sStoneType == -2) //-2 �Ի�ر��� �Ͼ �ű� ������ -> ���� �Ǵ� �ܺ� ���� ���ڷ� ����������Ѵ�
	{
		if(pstPrarticlePosCuda[tid].abExternalSide[0] || pstPrarticlePosCuda[tid].abExternalSide[1] || pstPrarticlePosCuda[tid].abExternalSide[2] || pstPrarticlePosCuda[tid].abExternalSide[3] || pstPrarticlePosCuda[tid].abExternalSide[4] || pstPrarticlePosCuda[tid].abExternalSide[5])
		{
			 //! �Ѱ��� ������ �ܺ� ����!
			pstPrarticlePosCuda[tid].sStoneType = -1;
		}
		else
		{
			
			pstPrarticlePosCuda[tid].sStoneType = 0; //! �Ѱ��� ������ ����
			//! �Ի�ر��� �Ͼ ������ �� ������ �����Լ����� 0.0���� �ʱ�ȭ
			pstPrarticlePosCuda[tid].fHaveWater = 0.0;
		}

		printf("%3d\t%3d\t%3d\t%d%d%d%d%d%d\t%d\tTid-%d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,
		pstPrarticlePosCuda[tid].abExternalSide[0],
		pstPrarticlePosCuda[tid].abExternalSide[1],
		pstPrarticlePosCuda[tid].abExternalSide[2],
		pstPrarticlePosCuda[tid].abExternalSide[3],
		pstPrarticlePosCuda[tid].abExternalSide[4],
		pstPrarticlePosCuda[tid].abExternalSide[5],
		pstPrarticlePosCuda[tid].sStoneType,
		tid);

	}
	else
	{
		//printf("%3d\t%3d\t%3d\t%d%d%d%d%d%d\t%d\tNo-%d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,
		//pstPrarticlePosCuda[tid].abExternalSide[0],
		//pstPrarticlePosCuda[tid].abExternalSide[1],
		//pstPrarticlePosCuda[tid].abExternalSide[2],
		//pstPrarticlePosCuda[tid].abExternalSide[3],
		//pstPrarticlePosCuda[tid].abExternalSide[4],
		//pstPrarticlePosCuda[tid].abExternalSide[5],
		//pstPrarticlePosCuda[tid].sStoneType,
		//tid);
	}

}


void CGPUCalcRockAgingInner::SetInnderVoxelData(int nRepeatCnt, int nPrarticlePosCnt, ST_PARTICLE_POS_CUDA	*pstPrarticlePos, ST_PARTICLE_POS_CUDA	*pstPrarticlePosMask)
{
	//! ���� ����
	ST_PARTICLE_POS_CUDA *pstPrarticlePosCuda;
	ST_PARTICLE_POS_CUDA *pstPrarticlePosCudaMask;
	//! ���� ����
	//int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS_CUDA);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}
	


	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}


	/*if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Memset \n" );
	}*/


	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}*/
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	if ( hipSuccess != hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice))
	{
		printf( "Error! Copy1 \n" );
	}

	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);

	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}
	
	//! �ּ� �����
	int nThreadCnt = 32; // 32�� ���� ó��
	//int nSharedMemoryCnt = lcm(nThreadCnt, 6); //92
	int nBlockCnt = (nPrarticlePosCnt / nThreadCnt) + 1;
	//kernelCalcRocking<<<nBlockCnt, nThreadCnt, nThreadCnt * 7>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	int nBlockStep = 256;
	
	for(int a = 0; a< nRepeatCnt; a++)
	{
		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			kernelCalcRocking<<<nBlockStepTemp, nThreadCnt>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, 
				m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, 
				m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, 
				m_fCalcWaterInnerAbsorption, 
				m_fCalcLayerWaterAborption, m_fCalcWaterChange);		
		}
		
		
		
		//nBlockStep = 256;
		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			
			kernelCalcRockingMasking<<<nBlockStepTemp, nThreadCnt>>>(nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask);
		}

		for(int n = 0; n < nBlockCnt; n += nBlockStep)
		{
			int nBlockStepTemp = nBlockStep;	
			if(nBlockCnt - n < nBlockStep)
				nBlockStepTemp = nBlockCnt - n;
			
			kernelReCalcExternalSide<<<nBlockStepTemp, nThreadCnt>>>(nPrarticlePosCnt, pstPrarticlePosCuda, m_nXFileVoxCnt, m_nYFileVoxCnt);
		}

	}
		

	

	//hipDeviceSynchronize();

	//hipDeviceSynchronize();
	//hipStreamSynchronize(cudaStream);


	//for(int n = 0; n < nPrarticlePosCnt; n++)
	//{
	//	printf("Acess idx : %d -> Porosity:%f,%f\n", n, pstPrarticlePosCudaMask[n].fPorosity, pstPrarticlePosCudaMask[n].fHaveWater);
	//}
	//printf("TEST\n");
	
	//memset(pstPrarticlePos, NULL, nSizeCnt*nPrarticlePosCnt);

	if ( hipSuccess != hipMemcpy(pstPrarticlePos, pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost))
	{
		printf( "Error! Copy2 \n" );
	}

	//hipDeviceSynchronize();

	hipFree(pstPrarticlePosCuda);
	hipFree(pstPrarticlePosCudaMask);
	//hipFree(pnPrarticlePosCntCuda);

	//pstPrarticlePosMask[0].x = 1;



}


