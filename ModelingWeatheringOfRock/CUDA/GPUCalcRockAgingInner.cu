#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"


CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}


//__global__ void kernelCalcRocking(int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
//								  int nX, int nY, int nZ,
//								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
//								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
//								  )
//{ 
//	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
//	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	int nExternalSideIdx = threadIdx.x;
//    //printf( "x:%d\n", nPrarticlePosCntCuda);
//
//	if(tid > nPrarticlePosCntCuda)
//		return;
//
//	
//    
//
//
//	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
//	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
//	{
//		__shared__ float			fPorosity;
//		fPorosity = 0.0;
//
//
//		if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == TRUE)
//		{
//			//switch(stParticlePos.vecExternalSide[nEx])	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			{
//			case 0:
//				{
//					fPorosity += (fCoefficient * fTopRate);
//				}
//				break;
//			case 1:
//				{
//					fPorosity += (fCoefficient * fBottomRate);
//				}
//				break;
//			case 2:
//			case 3:
//			case 4:
//			case 5:
//				{
//					fPorosity += (fCoefficient * fSideRate);
//				}
//				break;
//			default:
//				break;
//			}
//
//		}
//
//        __syncthreads();
//
//		pstPrarticlePosCudaMask[tid].fPorosity = fPorosity;
//		printf( "OuterMask : %d\n", tid);
//
//		//fPorosity ���ط� >= fGranularDisintegration �Ի�ر� ���ް�
//		//if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//		//{
//		//	
//
//		//	//stParticlePos.bUse = false;
//
//		//	//strKey.Format(L"%d-%d-%d",x,y,z);
//
//		//	//vecDeleParticle.push_back(strKey);
//
//		//	//break;
//		//}
//	}
//	else //! ���� ó��
//	{
//		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
//		{
//			printf( "InnerMask Check : %d\n", tid);
//
//
//			//__shared__ unsigned int sdata[6];
//
//			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
//			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
//			
//			//! 0���� ������ ������ ���� ������
//			if(fHaveWaterTemp < 0.0)
//				fHaveWaterTemp = 0.0;
//
//			__shared__ float			fPorosity[6];
//			memset(fPorosity, NULL, sizeof(float) * 6);
//			//fPorosity = 0.0;
//			//__syncthreads();
//
//			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
//			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
//			{
//				int nPos = tid;
//				switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//				{
//				case 0:
//					nPos += (nX * nY);
//					break;
//				case 1:
//					nPos -= (nX * nY);
//					break;
//				case 2:
//					nPos += nX;
//					break;
//				case 3:
//					nPos -= nX;
//					break;
//				case 4:
//					nPos -= 1;
//					break;
//				case 5:
//					nPos += 1;
//					break;
//				default:
//					break;
//				}
//
//				if(nPos > nPrarticlePosCntCuda)
//					return;
//
//				fPorosity[nExternalSideIdx] = (fCalcWaterChange / 5.0);
//
//				__syncthreads();
//
//				pstPrarticlePosCudaMask[nPos].fPorosity += fPorosity[nExternalSideIdx];
//
//				//if(pstPrarticlePosCudaMask[nPos].fPorosity >= pstPrarticlePosCudaMask[nPos].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//				//{
//				//	nBreakCnt++;
//
//				//	__syncthreads();
//				//	pstPrarticlePosCudaMask[nPos].fHaveWater = (pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) / (float)(nBreakCnt + 1);
//				//	printf( "InnerMask : %d\n", nPos);
//				//}
//			
//
//				//pstPrarticlePosCudaMask[nPos]
//
//			}
//
//		}
//	}
//
//	/*if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == true)
//	{
//
//	}*/
//
//
//	//pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx];
//
//    //printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);
//
//
//} 
//
//
//
//void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
//{
//	//! ���� ����
//	ST_PARTICLE_POS *pstPrarticlePosCuda;
//	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
//	//! ���� ����
//	//int *pnPrarticlePosCntCuda;
//
//	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
//	int nSizeCnt = sizeof(ST_PARTICLE_POS);
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//
//	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Memset \n" );
//	}
//
//	
//
//
//	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
//	{
//		printf( "Error! Malloc \n" );
//
//		Sleep(1000);
//	}*/
//	//else
//	//{
//	//	printf( "ErrorGOOD \n" );
//
//	//}
//
//
//	
//	
//	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
//	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
//	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);
//
//
//
//	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
//	//{
//	//	printf( "Error! Y VoxelState Memcpy \n" );
//	//}
//
//	
//	int nBlockCnt = nPrarticlePosCnt;
//	kernelCalcRocking<<<nBlockCnt, 6>>>(nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
//	
//
//	
//	hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost);
//
//
//	hipFree(pstPrarticlePosCuda);
//	hipFree(pstPrarticlePosCudaMask);
//	//hipFree(pnPrarticlePosCntCuda);
//
//	int a= 0;
//
//
//}


__global__ void kernelCalcRocking(int nThreadCnt, 
								  int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
								  int nXFileVoxCnt, int nYFileVoxCnt, int nZFileVoxCnt,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	 // ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nExternalSideIdx = threadIdx.x;
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid > nPrarticlePosCntCuda)
		return;
	
	__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[32 * 6];
	memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * 32 * 6);
	__syncthreads();

	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
	{
		//! �ܺ� �Ի�ر��� �������� ó���Ѵ�.(SHARED ����ŷ ���� �޸𸮿� ó��!)
		 astParticle_pos_unitProcess[threadIdx.x].fPorosity = (pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);
		
		//pstPrarticlePosCudaMask[tid].fPorosity = fPorosity;
		printf( "OuterMask : %d\n", tid);
	}
	else //! ���� ó��
	{
		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
		{
			printf( "InnerMask Check : %d\n", tid);


			//__shared__ unsigned int sdata[6];

			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
			
			//! 0���� ������ ������ ���� ������
			if(fHaveWaterTemp < 0.0)
				fHaveWaterTemp = 0.0;

			//fPorosity = 0.0;
			//__syncthreads();



			//(pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);


			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
			{
				int nPos = tid;
				switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
				{
				case 0:
					nPos += (nXFileVoxCnt * nYFileVoxCnt);
					break;
				case 1:
					nPos -= (nXFileVoxCnt * nYFileVoxCnt);
					break;
				case 2:
					nPos += nXFileVoxCnt;
					break;
				case 3:
					nPos -= nXFileVoxCnt;
					break;
				case 4:
					nPos -= 1;
					break;
				case 5:
					nPos += 1;
					break;
				default:
					break;
				}

				if(nPos > nPrarticlePosCntCuda)
					return;

				//fPorosity[nExternalSideIdx] = (fCalcWaterChange / 5.0);

				__syncthreads();

				pstPrarticlePosCudaMask[nPos].fPorosity += fPorosity[nExternalSideIdx];

				//if(pstPrarticlePosCudaMask[nPos].fPorosity >= pstPrarticlePosCudaMask[nPos].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
				//{
				//	nBreakCnt++;

				//	__syncthreads();
				//	pstPrarticlePosCudaMask[nPos].fHaveWater = (pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) / (float)(nBreakCnt + 1);
				//	printf( "InnerMask : %d\n", nPos);
				//}
			

				//pstPrarticlePosCudaMask[nPos]

			}

		}
	}

	/*if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == true)
	{

	}*/


	//pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx];

    //printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);


} 



void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
{
	//! ���� ����
	ST_PARTICLE_POS *pstPrarticlePosCuda;
	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
	//! ���� ����
	//int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}

	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}


	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Memset \n" );
	}

	


	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}*/
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	
	
	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);



	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}

	int nThreadCnt = 32;
	int nBlockCnt = (nPrarticlePosCnt / nThreadCnt) + 1;
	kernelCalcRocking<<<nBlockCnt, nThreadCnt * 6>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	

	
	hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost);


	hipFree(pstPrarticlePosCuda);
	hipFree(pstPrarticlePosCudaMask);
	//hipFree(pnPrarticlePosCntCuda);

	int a= 0;


}
