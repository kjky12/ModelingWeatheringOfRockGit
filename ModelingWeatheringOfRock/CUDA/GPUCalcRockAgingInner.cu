#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"


CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}


__global__ void kernelCalcRocking(int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid > nPrarticlePosCntCuda)
		return;

	

    printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);


} 



void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos)
{
	//! ���� ����
	ST_PARTICLE_POS *pstPrarticlePosCuda;
	//! ���� ����
	int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}

	if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	
	
	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
	hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);



	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}

	
	int nBlockCnt = nPrarticlePosCnt / 32;

	kernelCalcRocking<<<nBlockCnt, 32>>>(nPrarticlePosCnt, pstPrarticlePosCuda);
	//kernelCalcRocking<<<nBlockCnt, 32>>>(*pnPrarticlePosCntCuda, pstPrarticlePosCuda);



	hipFree(pstPrarticlePosCuda);
	hipFree(pnPrarticlePosCntCuda);

	int a= 0;


}

