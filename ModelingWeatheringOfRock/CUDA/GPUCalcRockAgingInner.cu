#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"

#define dfSHARED_MEMORY_SIZE	32*7

__device__ __host__ int gcd(int a, int b)
{
	int c;
	while (b != 0)
	{
		c = a % b;
		a = b;
		b = c;
	}
	return a;
}

__device__ __host__ int lcm(int a, int b)
{
    return a * b / gcd(a, b);
}




CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}


//__global__ void kernelCalcRocking(int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
//								  int nX, int nY, int nZ,
//								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
//								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
//								  )
//{ 
//	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
//	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	int nExternalSideIdx = threadIdx.x;
//    //printf( "x:%d\n", nPrarticlePosCntCuda);
//
//	if(tid > nPrarticlePosCntCuda)
//		return;
//
//	
//    
//
//
//	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
//	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
//	{
//		__shared__ float			fPorosity;
//		fPorosity = 0.0;
//
//
//		if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == TRUE)
//		{
//			//switch(stParticlePos.vecExternalSide[nEx])	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//			{
//			case 0:
//				{
//					fPorosity += (fCoefficient * fTopRate);
//				}
//				break;
//			case 1:
//				{
//					fPorosity += (fCoefficient * fBottomRate);
//				}
//				break;
//			case 2:
//			case 3:
//			case 4:
//			case 5:
//				{
//					fPorosity += (fCoefficient * fSideRate);
//				}
//				break;
//			default:
//				break;
//			}
//
//		}
//
//        __syncthreads();
//
//		pstPrarticlePosCudaMask[tid].fPorosity = fPorosity;
//		printf( "OuterMask : %d\n", tid);
//
//		//fPorosity ���ط� >= fGranularDisintegration �Ի�ر� ���ް�
//		//if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//		//{
//		//	
//
//		//	//stParticlePos.bUse = false;
//
//		//	//strKey.Format(L"%d-%d-%d",x,y,z);
//
//		//	//vecDeleParticle.push_back(strKey);
//
//		//	//break;
//		//}
//	}
//	else //! ���� ó��
//	{
//		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
//		{
//			printf( "InnerMask Check : %d\n", tid);
//
//
//			//__shared__ unsigned int sdata[6];
//
//			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
//			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
//			
//			//! 0���� ������ ������ ���� ������
//			if(fHaveWaterTemp < 0.0)
//				fHaveWaterTemp = 0.0;
//
//			__shared__ float			fPorosity[6];
//			memset(fPorosity, NULL, sizeof(float) * 6);
//			//fPorosity = 0.0;
//			//__syncthreads();
//
//			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
//			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
//			{
//				int nPos = tid;
//				switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
//				{
//				case 0:
//					nPos += (nX * nY);
//					break;
//				case 1:
//					nPos -= (nX * nY);
//					break;
//				case 2:
//					nPos += nX;
//					break;
//				case 3:
//					nPos -= nX;
//					break;
//				case 4:
//					nPos -= 1;
//					break;
//				case 5:
//					nPos += 1;
//					break;
//				default:
//					break;
//				}
//
//				if(nPos > nPrarticlePosCntCuda)
//					return;
//
//				fPorosity[nExternalSideIdx] = (fCalcWaterChange / 5.0);
//
//				__syncthreads();
//
//				pstPrarticlePosCudaMask[nPos].fPorosity += fPorosity[nExternalSideIdx];
//
//				//if(pstPrarticlePosCudaMask[nPos].fPorosity >= pstPrarticlePosCudaMask[nPos].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
//				//{
//				//	nBreakCnt++;
//
//				//	__syncthreads();
//				//	pstPrarticlePosCudaMask[nPos].fHaveWater = (pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) / (float)(nBreakCnt + 1);
//				//	printf( "InnerMask : %d\n", nPos);
//				//}
//			
//
//				//pstPrarticlePosCudaMask[nPos]
//
//			}
//
//		}
//	}
//
//	/*if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == true)
//	{
//
//	}*/
//
//
//	//pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx];
//
//    //printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);
//
//
//} 
//
//
//
//void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
//{
//	//! ���� ����
//	ST_PARTICLE_POS *pstPrarticlePosCuda;
//	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
//	//! ���� ����
//	//int *pnPrarticlePosCntCuda;
//
//	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
//	int nSizeCnt = sizeof(ST_PARTICLE_POS);
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Malloc \n" );
//	}
//
//
//	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
//	{
//		printf( "Error! Memset \n" );
//	}
//
//	
//
//
//	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
//	{
//		printf( "Error! Malloc \n" );
//
//		Sleep(1000);
//	}*/
//	//else
//	//{
//	//	printf( "ErrorGOOD \n" );
//
//	//}
//
//
//	
//	
//	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
//	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
//	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);
//
//
//
//	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
//	//{
//	//	printf( "Error! Y VoxelState Memcpy \n" );
//	//}
//
//	
//	int nBlockCnt = nPrarticlePosCnt;
//	kernelCalcRocking<<<nBlockCnt, 6>>>(nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
//	
//
//	
//	hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost);
//
//
//	hipFree(pstPrarticlePosCuda);
//	hipFree(pstPrarticlePosCudaMask);
//	//hipFree(pnPrarticlePosCntCuda);
//
//	int a= 0;
//
//
//}


__device__ void ViewMaskData(ST_PARTICLE_POS	*pstPrarticlePosCudaMask, int tid, int nPrarticlePosCntCuda, int nXFileVoxCnt, int nYFileVoxCnt)
{

	float f[7] = {pstPrarticlePosCudaMask[tid].fPorosity,0,0,0,0,0,0};
	float fW[7] = {pstPrarticlePosCudaMask[tid].fHaveWater,0,0,0,0,0,0};

	int nValue = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[1] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[1] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[2] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[2] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[3] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[3] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[4] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[4] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid - 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[5] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[5] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}

	nValue = tid + 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		f[6] = pstPrarticlePosCudaMask[nValue].fPorosity;
		fW[6] = pstPrarticlePosCudaMask[nValue].fHaveWater;
	}


	printf( "New Tid : %d -> Porosity:%f,%f,%f,%f,%f,%f,%f\nWater:%f,%f,%f,%f,%f,%f,%f\n", tid, 
		f[0],
		f[1],
		f[2],
		f[3],
		f[4],
		f[5],
		f[6],
		fW[0],
		fW[1],
		fW[2],
		fW[3],
		fW[4],
		fW[5],
		fW[6]
		);
    
}

__device__ void InputMaskDataToSharedMem(ST_PARTICLE_POS	*pstPrarticlePosCudaMask, ST_PARTICLE_POS_UNIT_PROCESS	*astParticle_pos_unitProcess, int tid, int threadIdx, int nPrarticlePosCntCuda, int nXFileVoxCnt, int nYFileVoxCnt)
{

	pstPrarticlePosCudaMask[tid].fPorosity += astParticle_pos_unitProcess[threadIdx].fPorosity;
	pstPrarticlePosCudaMask[tid].fHaveWater += astParticle_pos_unitProcess[threadIdx].fHaveWater;


	int nValue = tid + (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 1].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 1].fHaveWater;

	

	}

	nValue = tid - (nXFileVoxCnt * nYFileVoxCnt);
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{

		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 2].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 2].fHaveWater;

	
	}

	nValue = tid + nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 3].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 3].fHaveWater;

		
	}

	nValue = tid - nXFileVoxCnt;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 4].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 4].fHaveWater;

	}

	nValue = tid - 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 5].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 5].fHaveWater;


	}

	nValue = tid + 1;
	if(nValue >= 0 && nValue < nPrarticlePosCntCuda)
	{
		pstPrarticlePosCudaMask[nValue].fPorosity += astParticle_pos_unitProcess[threadIdx+ 6].fPorosity;
		pstPrarticlePosCudaMask[nValue].fHaveWater += astParticle_pos_unitProcess[threadIdx+ 6].fHaveWater;
	}

	
	

}


__global__ void kernelCalcRocking(int nThreadCnt, 
								  int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
								  int nXFileVoxCnt, int nYFileVoxCnt, int nZFileVoxCnt,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	//return;

	 // ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//if(tid > nPrarticlePosCntCuda)
	//	return;

	
	//return;
	//int nExternalSideIdx = threadIdx.x % 6;
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid > nPrarticlePosCntCuda)
		return;

	//printf("tid : %d\n", tid);

	//extern __shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	//__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[32 * 7]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	__shared__ ST_PARTICLE_POS_UNIT_PROCESS			astParticle_pos_unitProcess[dfSHARED_MEMORY_SIZE]; //->0:����, 1:��,2:��,3:��,4:��,5:��,6��
	memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * dfSHARED_MEMORY_SIZE);
	//memset(astParticle_pos_unitProcess, NULL, sizeof(ST_PARTICLE_POS_UNIT_PROCESS) * lcm(nThreadCnt, 6));
	__syncthreads();

	//ViewMaskData(pstPrarticlePosCudaMask, tid, nPrarticlePosCntCuda, nXFileVoxCnt, nYFileVoxCnt);


	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
	{
		//! �ܺ� �Ի�ر��� �������� ó���Ѵ�.(SHARED ����ŷ ���� �޸𸮿� ó��!)
		 astParticle_pos_unitProcess[threadIdx.x * 7].fPorosity = (pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			 +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);
		
		
		//printf( "OuterMask : %d -> Porosity:%f\n", tid, astParticle_pos_unitProcess[threadIdx.x].fPorosity);
	}
	else //! ���� ó��
	{
		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
		{
			//printf( "InnerMask Check : %d\n", tid);


			//__shared__ unsigned int sdata[6];

			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
			
			//! 0���� ������ ������ ���� ������
			if(fHaveWaterTemp < 0.0)
				fHaveWaterTemp = 0.0;

			//fPorosity = 0.0;
			//__syncthreads();

			//! ����ŷ��
			astParticle_pos_unitProcess[threadIdx.x].fHaveWater += fHaveWaterTemp;

			//printf( "Inner : %d -> Water:%f\n", tid, astParticle_pos_unitProcess[threadIdx.x].fHaveWater);

			//(pstPrarticlePosCuda[tid].abExternalSide[0] * fCoefficient * fTopRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[1] * fCoefficient * fBottomRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[2] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[3] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[4] * fCoefficient * fSideRate)
			// +  (pstPrarticlePosCuda[tid].abExternalSide[5] * fCoefficient * fSideRate);


			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
			{
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity += (fCalcWaterChange / 5.0);
				//! ��
				astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity += (fCalcWaterChange / 5.0);

			}

		}
	}



	pstPrarticlePosCudaMask[tid].fPorosity += astParticle_pos_unitProcess[threadIdx.x].fPorosity;
	pstPrarticlePosCudaMask[tid].fHaveWater += astParticle_pos_unitProcess[threadIdx.x].fHaveWater;
	
	pstPrarticlePosCudaMask[tid + (nXFileVoxCnt * nYFileVoxCnt)].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 1].fPorosity;
	pstPrarticlePosCudaMask[tid + (nXFileVoxCnt * nYFileVoxCnt)].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 1].fHaveWater;

	pstPrarticlePosCudaMask[tid - (nXFileVoxCnt * nYFileVoxCnt)].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 2].fPorosity;
	pstPrarticlePosCudaMask[tid - (nXFileVoxCnt * nYFileVoxCnt)].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 2].fHaveWater;

	pstPrarticlePosCudaMask[tid + nXFileVoxCnt].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 3].fPorosity;
	pstPrarticlePosCudaMask[tid + nXFileVoxCnt].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 3].fHaveWater;

	pstPrarticlePosCudaMask[tid - nXFileVoxCnt].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 4].fPorosity;
	pstPrarticlePosCudaMask[tid - nXFileVoxCnt].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 4].fHaveWater;

	pstPrarticlePosCudaMask[tid - 1].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 5].fPorosity;
	pstPrarticlePosCudaMask[tid - 1].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 5].fHaveWater;

	pstPrarticlePosCudaMask[tid + 1].fPorosity += astParticle_pos_unitProcess[threadIdx.x + 6].fPorosity;
	pstPrarticlePosCudaMask[tid + 1].fHaveWater += astParticle_pos_unitProcess[threadIdx.x + 6].fHaveWater;
	


} 


void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
{
	//! ���� ����
	ST_PARTICLE_POS *pstPrarticlePosCuda;
	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
	//! ���� ����
	//int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}

	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}


	/*if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Memset \n" );
	}*/


	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}*/
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	
	
	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);



	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}
	
	//! �ּ� �����
	int nThreadCnt = 32; // 32�� ���� ó��
	//int nSharedMemoryCnt = lcm(nThreadCnt, 6); //92
	int nBlockCnt = (nPrarticlePosCnt / nThreadCnt) + 1;
	//kernelCalcRocking<<<nBlockCnt, nThreadCnt, nThreadCnt * 7>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	kernelCalcRocking<<<nBlockCnt, nThreadCnt>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	//kernelCalcRocking<<<4, nThreadCnt, nThreadCnt * 7>>>(nThreadCnt, nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	
	//hipDeviceSynchronize();
	//hipStreamSynchronize(cudaStream);

	printf("TEST\n");
	
	if ( hipSuccess != hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost))
	{
		printf( "Error! Copy \n" );
	}


	hipFree(pstPrarticlePosCuda);
	hipFree(pstPrarticlePosCudaMask);
	//hipFree(pnPrarticlePosCntCuda);

	//pstPrarticlePosMask[0].x = 1;


	int a= 0;


}


