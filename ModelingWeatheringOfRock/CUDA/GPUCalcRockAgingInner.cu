#include "hip/hip_runtime.h"
#include "GPUCalcRockAgingInner.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "Windows.h"


CGPUCalcRockAgingInner::CGPUCalcRockAgingInner(void)
{

}

CGPUCalcRockAgingInner::~CGPUCalcRockAgingInner(void)
{

}


__global__ void kernelCalcRocking(int nPrarticlePosCntCuda, ST_PARTICLE_POS	*pstPrarticlePosCuda, ST_PARTICLE_POS	*pstPrarticlePosCudaMask, 
								  int nX, int nY, int nZ,
								  float fCoefficient, float fTopRate, float fSideRate, float fBottomRate,
								  float fCalcWaterInnerAbsorption, float fCalcLayerWaterAborption, float fCalcWaterChange
								  )
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int nExternalSideIdx = threadIdx.x;
    //printf( "x:%d\n", nPrarticlePosCntCuda);

	if(tid > nPrarticlePosCntCuda)
		return;

	
    


	//printf( "IN_OUT : %d\n", pstPrarticlePosCuda[tid].bInOut);
	if(pstPrarticlePosCuda[tid].bInOut == true) //! �ܺδ� ���� �Ի�ر�
	{
		__shared__ float			fPorosity;
		fPorosity = 0.0;


		if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == TRUE)
		{
			//switch(stParticlePos.vecExternalSide[nEx])	//[0:��,1:��,2:��,3:��,4:��,5:��]
			switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
			{
			case 0:
				{
					fPorosity += (fCoefficient * fTopRate);
				}
				break;
			case 1:
				{
					fPorosity += (fCoefficient * fBottomRate);
				}
				break;
			case 2:
			case 3:
			case 4:
			case 5:
				{
					fPorosity += (fCoefficient * fSideRate);
				}
				break;
			default:
				break;
			}

		}

        __syncthreads();

		pstPrarticlePosCudaMask[tid].fPorosity = fPorosity;
		printf( "OuterMask : %d\n", tid);

		//fPorosity ���ط� >= fGranularDisintegration �Ի�ر� ���ް�
		//if(pstPrarticlePosCuda[tid].fPorosity >= pstPrarticlePosCuda[tid].fGranularDisintegration) // �Ի�ر� ���ް��� �����Ͽ� ����
		//{
		//	

		//	//stParticlePos.bUse = false;

		//	//strKey.Format(L"%d-%d-%d",x,y,z);

		//	//vecDeleParticle.push_back(strKey);

		//	//break;
		//}
	}
	else //! ���� ó��
	{
		if(pstPrarticlePosCuda[tid].sStoneType == 0) //! ���ظ� ó���ؾ��Ѵ�.
		{
			printf( "InnerMask Check : %d\n", tid);


			//__shared__ unsigned int sdata[6];

			//1. ���� ����� = ��������� - {(�ִ� ���̾� Idx - ���� ���̾� Idx) * ���̾ ���� ������ * ��������� }
			float fHaveWaterTemp = fCalcWaterInnerAbsorption - ( (/*sMaxLayerIdx -*/ pstPrarticlePosCuda[tid].sLayerIdx) * fCalcLayerWaterAborption * fCalcWaterInnerAbsorption );
			
			//! 0���� ������ ������ ���� ������
			if(fHaveWaterTemp < 0.0)
				fHaveWaterTemp = 0.0;

			__shared__ int			nBreakCnt;
			nBreakCnt = 0;

			//3. if(���� ��ȭ�� * ���� ��â�� > 1.0)
			if(((pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) * fCalcWaterChange) > 1.0)
			{
				int nPos = tid;
				switch(nExternalSideIdx)	//[0:��,1:��,2:��,3:��,4:��,5:��]
				{
				case 0:
					nPos += (nX * nY);
					break;
				case 1:
					nPos -= (nX * nY);
					break;
				case 2:
					nPos += nX;
					break;
				case 3:
					nPos -= nX;
					break;
				case 4:
					nPos -= 1;
					break;
				case 5:
					nPos += 1;
					break;
				default:
					break;
				}

				if(nPos > nPrarticlePosCntCuda)
					return;

				nBreakCnt++;

				__syncthreads();


				pstPrarticlePosCudaMask[nPos].fHaveWater = (pstPrarticlePosCuda[tid].fHaveWater + fHaveWaterTemp) / (float)(nBreakCnt + 1);
				printf( "InnerMask : %d\n", nPos);
				//pstPrarticlePosCudaMask[nPos]

			}

		}
	}

	/*if(pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx] == true)
	{

	}*/


	//pstPrarticlePosCuda[tid].abExternalSide[nExternalSideIdx];

    //printf( "x:%d\ty:%d\tz:%d\tTid : %d\n", pstPrarticlePosCuda[tid].x, pstPrarticlePosCuda[tid].y, pstPrarticlePosCuda[tid].z,  tid);


} 



void CGPUCalcRockAgingInner::SetInnderVoxelData(int nPrarticlePosCnt, ST_PARTICLE_POS	*pstPrarticlePos, ST_PARTICLE_POS	*pstPrarticlePosMask)
{
	//! ���� ����
	ST_PARTICLE_POS *pstPrarticlePosCuda;
	ST_PARTICLE_POS *pstPrarticlePosCudaMask;
	//! ���� ����
	//int *pnPrarticlePosCntCuda;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	int nSizeCnt = sizeof(ST_PARTICLE_POS);
	if ( hipSuccess != hipMalloc(&pstPrarticlePosCuda, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}

	if ( hipSuccess != hipMalloc(&pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Malloc \n" );
	}


	if ( hipSuccess != hipMemset(pstPrarticlePosCudaMask, NULL, nSizeCnt*nPrarticlePosCnt))
	{
		printf( "Error! Memset \n" );
	}

	


	/*if( hipSuccess != hipMalloc(&pnPrarticlePosCntCuda, sizeof(int)))
	{
		printf( "Error! Malloc \n" );

		Sleep(1000);
	}*/
	//else
	//{
	//	printf( "ErrorGOOD \n" );

	//}


	
	
	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(pstPrarticlePosCuda, pstPrarticlePos, nSizeCnt*nPrarticlePosCnt, hipMemcpyHostToDevice);
	//hipMemcpy(pnPrarticlePosCntCuda, &nPrarticlePosCnt, sizeof(int), hipMemcpyHostToDevice);



	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}

	
	int nBlockCnt = nPrarticlePosCnt;

	

	kernelCalcRocking<<<nBlockCnt, 6>>>(nPrarticlePosCnt, pstPrarticlePosCuda, pstPrarticlePosCudaMask, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, m_fCoefficient, m_fTopRate, m_fSideRate, m_fBottomRate, m_fCalcWaterInnerAbsorption, m_fCalcLayerWaterAborption, m_fCalcWaterChange);
	//kernelCalcRocking<<<nBlockCnt, 32>>>(*pnPrarticlePosCntCuda, pstPrarticlePosCuda);

	
	hipMemcpy(pstPrarticlePosMask, pstPrarticlePosCudaMask, nSizeCnt*nPrarticlePosCnt, hipMemcpyDeviceToHost);


	hipFree(pstPrarticlePosCuda);
	hipFree(pstPrarticlePosCudaMask);
	//hipFree(pnPrarticlePosCntCuda);

	int a= 0;


}

