#include "hip/hip_runtime.h"
#include "GPUSolid.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>


#define SIZE 1024

CGPUSolid::CGPUSolid(void)
{

}

CGPUSolid::~CGPUSolid(void)
{

}




__global__ void sum_Kernel(int a, int b, int* c)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	c[tid] = a + b;
}



int CGPUSolid::sum_cuda(int a, int b, int *c)
{
	int *f;

	hipMalloc(&f, sizeof(int) * 1);
	hipMemcpy(f, c, sizeof(int) * 1, hipMemcpyHostToDevice);

	sum_Kernel << <1, 1>> > (a,b, f);

	hipMemcpy(c, f, sizeof(int) * 1, hipMemcpyDeviceToHost);

	hipFree(f);


	return true;
}






__global__ void VectorAdd(int *a, int *b, int *c, int n)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	int i = threadIdx.x; 
	printf("threadIdx.x : %d, n : %d\n", i, n); 
	for (i = 0; i < n; i++) 
	{ 
		c[i] = a[i] + b[i]; 
		printf("%d = %d + %d\n", c[i], a[i], b[i]); 
	} 
} 



int CGPUSolid::Sum_TEST(int *a, int *b, int *c)
{
	int *d_a, *d_b, *d_c;

	// hipMalloc(destination, number of byte)�� device�� �޸𸮸� �Ҵ��Ѵ�.
	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));
	
	
	// hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)�� ȣ��Ʈ���� ����̽��� �޸𸮸� ī���Ѵ�.
	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);
	
	// �Լ� ȣ���� ���ؼ� ���ο� ���ؽ� ��Ҹ� �߰��� �ʿ䰡 �ִ�.
	// ù��° parameter�� ���� ���̴�. ���������� ������ ���� �ϳ��̴�.
	// SIZE�� 1024���� �����带 �ǹ��Ѵ�.
	VectorAdd << < 1, SIZE >> >(d_a, d_b, d_c, SIZE);
	
	
	//hipMemcpy(source, destination, number of byte, cudaMemDeviceToHost)�� ����̽��� �޸�(���� ��� ������)�� ȣ��Ʈ�� ī���Ѵ�.
	hipMemcpy(a, d_a, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, d_b, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);
	

	for (int i = 0; i<SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	// hipFree(d_a)�� ���� ����̽��� �޸𸮸� �Ҵ� ����
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
return 0;
}


void CGPUSolid::GpuInfoPrint()
{
	hipDeviceProp_t  prop;

    int count = 0;
    hipGetDeviceCount( &count );

    for (int i=0; i< count; i++) 
	{
        hipGetDeviceProperties( &prop, i );
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );
		printf( "\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );
		printf( "\n" );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n", prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
        printf( "\n" );


		m_nMaxBlockSize = prop.maxGridSize[0];
		if(m_nMaxBlockSize > prop.maxGridSize[1])
			m_nMaxBlockSize = prop.maxGridSize[1];
		if(m_nMaxBlockSize > prop.maxGridSize[2])
			m_nMaxBlockSize = prop.maxGridSize[2];

		m_nMaxThreadSize = prop.maxThreadsPerBlock ;
    }

    return ;

}



__global__ void UseCheckX(bool *bValue, bool *bResult, int nSize)
{ 
	// ������ �����尡 ���ÿ� ó���Ѵ�. // ���� threadIdx(������ �ε���)�� ���ؼ� ��������� �����Ѵ�. 
	//int i = threadIdx.x; 
	//printf("threadIdx.x : %d, n : %d\n", i, n); 
	//for (i = 0; i < n; i++) 
	//{ 
	//	c[i] = a[i] + b[i]; 
	//	printf("%d = %d + %d\n", c[i], a[i], b[i]); 
	//} 

	int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int nPos = tid * nSize;

	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos);


	int nOutCnt = 0;
	int nStartIdx = -1, nEndIdx = -1;
	int x = 0;
	for(x = 0 ; x < nSize ; x++)
	{
		if(bValue[nPos + x])	// ù �����͸� ã�´�.
		{
			nStartIdx = x;
			break;
		}
		else
			nOutCnt++;
	}

	if(nSize != nOutCnt)
	{
		for(x = nSize-1 ; x >=0  ; x--)
		{
			if(bValue[nPos + x])	// ù �����͸� ã�´�.
			{
				nEndIdx = x;
				break;
			}
		}

		
	}

	if(nStartIdx != -1 && nEndIdx != -1)
	{
		//printf("S : %d\t, E : %d\n", nStartIdx, nEndIdx );
		for(unsigned int s = nStartIdx; s <= nEndIdx ; s++)
		{
			
			bResult[nPos + s] = 1;
			//pParent->ReplaceFileInfo(s,y,z,1);
		}
	}


	//c[tid] = a[tid] + b[tid];
} 


void CGPUSolid::CUDA_SolidVoxelizationX(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelXResult)
{
	//m_nMaxBlockSize = 15;
	//m_nMaxThreadSize = 16;

	const unsigned int unOneThreadSize = m_nXFileVoxCnt;
	const unsigned int unBlockSize = m_nYFileVoxCnt * m_nZFileVoxCnt;
	unsigned int unOneStep  = m_nMaxBlockSize * unOneThreadSize;
	if(unOneStep > unBlockSize * unOneThreadSize)
		unOneStep = unBlockSize * unOneThreadSize;
			
	unsigned int unStep = 0;

	while (unStep < unVoxelSize)
	{
		unsigned int unCpySize = 0;
		//! ����̽����� �޸� ī��
		if(unVoxelSize - unStep < unOneStep)
		{
			printf("OH!");
			unCpySize = unVoxelSize - unStep;
		}
		else
			unCpySize = unOneStep;

		bool *cubVoxelState;
		bool *cubVoxelResultSize;

		hipMalloc(&cubVoxelState, unCpySize*sizeof(bool));
		hipMalloc(&cubVoxelResultSize, unCpySize*sizeof(bool));

		//! ����̽����� �޸� ī��
		hipMemcpy(cubVoxelState, bVoxelState + unStep, unCpySize*sizeof(bool), hipMemcpyHostToDevice);
		
		UseCheckX<<<unCpySize / unOneThreadSize,1>>>(cubVoxelState, cubVoxelResultSize, unOneThreadSize);


		hipMemcpy(bVoxelXResult + unStep, cubVoxelResultSize, unCpySize*sizeof(bool), hipMemcpyDeviceToHost);

		unStep += unCpySize;

		hipFree(cubVoxelState);
		hipFree(cubVoxelResultSize);

	}

	//return true;
}


__global__ void UseCheckY(bool *bValue, bool *bResult, int nX, int nY, int nZ, int unBlockStep)
{ 
	int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int nDiv = 0;
	int nNmg = 0;
	int nStad = nX;

	int nPos = 0;
	if((unBlockStep + tid) > 0)
	{
		nDiv = (unBlockStep + tid) / nStad;
		nNmg = (unBlockStep + tid) % nStad;
	}
	//nPos = tid + nDiv * (nStad * nY)  + nNmg;
	nPos = nDiv * (nStad * nY)  + nNmg;

	int nOutCnt = 0;
	int nStartIdx = -1, nEndIdx =-1;
	int nIdx = 0;
	for(nIdx = 0 ; nIdx < nY ; nIdx++)
	{
		//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos + (nIdx * nX));
		//printf("tid : %02d,\tPos : %02d,\tIdx*nStad : %02d,\tIdx %02d\n", tid, nPos, (nIdx * nStad), nPos + (nIdx * nStad));
		if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
		{
			nStartIdx = nIdx * nStad;
			break;
		}
		else
			nOutCnt++;
	}

	if(nY != nOutCnt)
	{
		for(nIdx = nY-1 ; nIdx >=0  ; nIdx--)
		{
			//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos + (nIdx * nX));

			//printf("tid : %02d,\tPos : %02d,\tIdx*nStad : %02d,\tIdx %02d\n", tid, nPos, (nIdx * nStad), nPos + (nIdx * nStad));
			if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
			{
				nEndIdx = nIdx * nStad;
				break;
			}
		}
	}

	if(nStartIdx != -1 && nEndIdx != -1)
	{
		//printf("tid: %03d,\t S : %d\t, E : %d\t, OutCnt : %d\n", tid, nStartIdx, nEndIdx, nOutCnt );
		for(unsigned int s = nStartIdx; s <= nEndIdx ; s= s + nStad)
		{
			//printf("%d\t", s);
			bResult[nPos + s] = 1;
			//pParent->ReplaceFileInfo(s,y,z,1);
		}
	}
	

	
} 

void CGPUSolid::CUDA_SolidVoxelizationY(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelYResult)
{
	//m_nMaxBlockSize = 15;
	//m_nMaxThreadSize = 16;

	const unsigned int unOneThreadSize = m_nYFileVoxCnt;
	const unsigned int unMaxBlockCnt = m_nMaxBlockSize;
	unsigned int unOneStepBlcokCnt  = m_nXFileVoxCnt * m_nZFileVoxCnt;
	if(unOneStepBlcokCnt > unMaxBlockCnt)
	{
		unOneStepBlcokCnt = unMaxBlockCnt;
	}
			
	const unsigned int unMaxBlockSize = unVoxelSize / m_nYFileVoxCnt;

	unsigned int unBlockStep = 0;

	bool *cubVoxelState;
	bool *cubVoxelResultSize;

	if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Y VoxelState Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultSize, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Y VoxelResult Malloc \n" );
	}

	//! ����̽����� �޸� ī��
	hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	while (unBlockStep <  unMaxBlockSize)
	{
		if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
			unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;
		UseCheckY<<<unOneStepBlcokCnt,1>>>(cubVoxelState, cubVoxelResultSize, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! Y BlockStep : %d \n", unBlockStep);
		}
		unBlockStep += unOneStepBlcokCnt;
		
	}
	
	if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	{
		printf( "Error! Y VoxelState Memcpy \n" );
	}

	hipFree(cubVoxelState);
	hipFree(cubVoxelResultSize);
}


__global__ void UseCheckZ(bool *bValue, bool *bResult, int nX, int nY, int nZ, int unBlockStep)
{ 
	int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	int nDiv = 0;
	int nNmg = 0;
	int nStad = nX*nY;

	int nPos = 0;
	if((unBlockStep + tid) > 0)
	{
		nDiv = (unBlockStep + tid) / nStad;
		nNmg = (unBlockStep + tid) % nStad;
	}
	//nPos = tid + nDiv * (nStad * nZ)  + nNmg;
	nPos = nDiv * (nStad * nZ)  + nNmg;

	int nOutCnt = 0;
	int nStartIdx = -1, nEndIdx =-1;
	int nIdx = 0;
	for(nIdx = 0 ; nIdx < nZ ; nIdx++)
	{
		//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos + (nIdx * (nX*nY)));
		//printf("tid : %02d,\tPos : %02d,\tIdx*nStad : %02d,\tIdx %02d\n", tid, nPos, (nIdx * nStad), nPos + (nIdx * nStad));
		if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
		{
			nStartIdx = nIdx * nStad;
			break;
		}
		else
			nOutCnt++;
	}

	if(nZ != nOutCnt)
	{
		for(nIdx = nZ-1 ; nIdx >=0  ; nIdx--)
		{
			//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos + (nIdx * (nX*nY)));

			//printf("tid : %02d,\tPos : %02d,\tIdx*nStad : %02d,\tIdx %02d\n", tid, nPos, (nIdx * nStad), nPos + (nIdx * nStad));
			if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
			{
				nEndIdx = nIdx * nStad;
				break;
			}
		}
	}

	//printf("tid: %03d,\t S : %d\t, E : %d\t, OutCnt : %d\n", tid, nStartIdx, nEndIdx, nOutCnt );
	if(nStartIdx != -1 && nEndIdx != -1)
	{
		//printf("tid: %03d,\t S : %d\t, E : %d\t, OutCnt : %d\n", tid, nStartIdx, nEndIdx, nOutCnt );
		for(unsigned int s = nStartIdx; s <= nEndIdx ; s= s + nStad)
		{
			//printf("%d\t", s);
			bResult[nPos + s] = 1;
			//pParent->ReplaceFileInfo(s,y,z,1);
		}
	}

	
} 

void CGPUSolid::CUDA_SolidVoxelizationZ(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelZResult)
{
	//m_nMaxBlockSize = 15;
	//m_nMaxThreadSize = 16;

	const unsigned int unOneThreadSize = m_nZFileVoxCnt;
	const unsigned int unMaxBlockCnt = m_nMaxBlockSize;	
	unsigned int unOneStepBlcokCnt  = m_nXFileVoxCnt * m_nYFileVoxCnt;
	//unsigned int unOneStepBlcokCnt  = 1;
	if(unOneStepBlcokCnt > unMaxBlockCnt)
	{
		unOneStepBlcokCnt = unMaxBlockCnt;
	}

	const unsigned int unMaxBlockSize = unVoxelSize / m_nZFileVoxCnt;

	unsigned int unBlockStep = 0;

	bool *cubVoxelState;
	bool *cubVoxelResultSize;

	if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Z VoxelState Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultSize, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Z VoxelResult Malloc \n" );
	}

	//! ����̽����� �޸� ī��
	hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	while (unBlockStep <  unMaxBlockSize)
	{
		if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
			unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;
		UseCheckZ<<<unOneStepBlcokCnt,1>>>(cubVoxelState, cubVoxelResultSize, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! Z BlockStep : %d \n", unBlockStep);
		}
		unBlockStep += unOneStepBlcokCnt;
	}
	


	if ( hipSuccess != hipMemcpy(bVoxelZResult, cubVoxelResultSize, unVoxelSize*sizeof(bool)*sizeof(bool), hipMemcpyDeviceToHost))
	{
		printf( "Error! Z VoxelState Memcpy \n" );
	}


	hipFree(cubVoxelState);
	hipFree(cubVoxelResultSize);
}


void CGPUSolid::CUDA_SolidVoxelizationZTest(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelZResult, unsigned int unBlockStep)
{
	//m_nMaxBlockSize = 15;
	//m_nMaxThreadSize = 16;

	const unsigned int unMaxBlockCnt = m_nMaxBlockSize;	
	//unsigned int unOneStepBlcokCnt  = m_nXFileVoxCnt * m_nYFileVoxCnt;
	unsigned int unOneStepBlcokCnt  = 1;
	if(unOneStepBlcokCnt > unMaxBlockCnt)
	{
		unOneStepBlcokCnt = unMaxBlockCnt;
	}

	const unsigned int unMaxBlockSize = unVoxelSize / m_nZFileVoxCnt;


	bool *cubVoxelState;
	bool *cubVoxelResultSize;

	if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Z VoxelState Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultSize, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Z VoxelResult Malloc \n" );
	}

	//! ����̽����� �޸� ī��
	hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);
	hipMemcpy(cubVoxelResultSize, bVoxelZResult, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	//while (unBlockStep <  unMaxBlockSize)
	//{
		//if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
		//	unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;
		UseCheckZ<<<unOneStepBlcokCnt,1>>>(cubVoxelState, cubVoxelResultSize, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! Z BlockStep : %d \n", unBlockStep);
		}
	//unBlockStep += unOneStepBlcokCnt;
	//}
	

	


	if ( hipSuccess != hipMemcpy(bVoxelZResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	{
		printf( "Error! Z VoxelState Memcpy \n" );
	}

	//for (int v= 0; v < unVoxelSize; v ++)
	//{
	//	printf("%d\n", bVoxelZResult[v]);
	//}


	hipFree(cubVoxelState);
	hipFree(cubVoxelResultSize);
}

__global__ void MergeXYZBig(bool *bVoxelStateX, bool *bVoxelStateY, bool *bVoxelStateZ, bool *bVoxelStateTotal, unsigned int unOneStep)
{ 
	//printf("gogogog\n");
	unsigned int  tid = blockDim.x * blockIdx.x + threadIdx.x;// + unStep;
	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\tthreadIdx.y :%d\t tid : %02d\n", blockIdx.x, blockDim.x, threadIdx.x, threadIdx.y,tid);
	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02ld\n", blockIdx.x, blockDim.x, threadIdx.x, tid);


	if(unOneStep > tid)
	{
		//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\tthreadIdx.y :%d\t tid : %02d\n", blockIdx.x, blockDim.x, threadIdx.x, threadIdx.y,tid);

		/*__shared__ bool b[3];
		b[0] = bVoxelStateX[tid];
		b[1] = bVoxelStateY[tid];
		b[2] = bVoxelStateZ[tid];*/

		//if(bVoxelStateX[tid] && bVoxelStateY[tid] && bVoxelStateZ[tid])
		//	bVoxelStateTotal[tid] = true;

		bVoxelStateTotal[tid] = bVoxelStateX[tid] & bVoxelStateY[tid] & bVoxelStateZ[tid];


		//if(bVoxelStateX[tid] && bVoxelStateY[tid] && bVoxelStateZ[tid])
		//	bVoxelStateTotal[tid] = true;

	}
	
	


} 

void CGPUSolid::CUDA_SolidVoxelizationTotal(unsigned int  unVoxelSize, bool *bVoxelStateX, bool *bVoxelStateY, bool *bVoxelStateZ, bool *bVoxelStateTotal)
{
	//m_nMaxBlockSize = 32;
	//m_nMaxThreadSize = 32;

	unsigned int unOneStep  = m_nMaxBlockSize * m_nMaxThreadSize;
	if(unOneStep > unVoxelSize)
		unOneStep = unVoxelSize;
			
	unsigned int unStep = 0;
	unsigned int unIdx = 0;

	while (unStep < unVoxelSize)
	{
		unsigned int unCpySize = 0;
		//! ����̽����� �޸� ī��
		if(unVoxelSize - unStep < unOneStep)
			unCpySize = unVoxelSize - unStep;
		else
			unCpySize = unOneStep;

		bool *cubVoxelStateX;
		bool *cubVoxelStateY, 
		bool *cubVoxelStateZ;
		bool *cubVoxelResultSize;
		hipMalloc(&cubVoxelStateX, unCpySize*sizeof(bool));
		hipMalloc(&cubVoxelStateY, unCpySize*sizeof(bool));
		hipMalloc(&cubVoxelStateZ, unCpySize*sizeof(bool));
		hipMalloc(&cubVoxelResultSize, unCpySize*sizeof(bool));

		

		hipMemcpy(cubVoxelStateX, bVoxelStateX + (unOneStep*unIdx), unCpySize*sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(cubVoxelStateY, bVoxelStateY + (unOneStep*unIdx), unCpySize*sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(cubVoxelStateZ, bVoxelStateZ + (unOneStep*unIdx), unCpySize*sizeof(bool), hipMemcpyHostToDevice);
		
		//printf("%d\n", unStep);
		MergeXYZBig<<<m_nMaxBlockSize,m_nMaxThreadSize>>>(cubVoxelStateX, cubVoxelStateY, cubVoxelStateZ, cubVoxelResultSize, unCpySize);

		hipMemcpy(bVoxelStateTotal + (unOneStep*unIdx), cubVoxelResultSize, unCpySize*sizeof(bool), hipMemcpyDeviceToHost);

		unStep += unCpySize;
		unIdx += 1;

		hipFree(cubVoxelStateX);
		hipFree(cubVoxelStateY);
		hipFree(cubVoxelStateZ);
		hipFree(cubVoxelResultSize);
	}

	



	//return true;
}




__global__ void UseCheckXYZ(bool *bValue, bool *bResultX, bool *bResultY, bool *bResultZ, int nX, int nY, int nZ, int unBlockStep)
{ 
	unsigned int  tid = blockDim.x * blockIdx.x / 3;/* + threadIdx.x;*/// + unStep;

	int nDiv = 0;
	int nNmg = 0;
	//! �� �ະ�� ���� ù IDx + ?? �� ����..?
	int nStad = 0;
	//! ��� �� ���� ������ �־���
	int nX_Y_Z = 0;
	bool *bResult = NULL;

	if(threadIdx.x == 0) // X��
	{
		//nStad = nX;
		nStad = 1;
		nX_Y_Z = nX;
		bResult = bResultX;

		if(nY * nZ <= tid)
			return;
	}
	else if(threadIdx.x == 1) // Y��
	{
		nStad = nX;
		nX_Y_Z = nY;
		bResult = bResultY;

		if(nX * nZ <= tid)
			return;
	}
	else //if(threadIdx.x == 2) Z��
	{
		nStad = nX*nY;
		nX_Y_Z = nZ;
		bResult = bResultZ;
		
		if(nX * nY <= tid)
			return;
	}


	//! �Ʒ��� ���..
	if((unBlockStep + tid) > 0)
	{
		nDiv = (unBlockStep + tid) / nStad;
		nNmg = (unBlockStep + tid) % nStad;
	}
	int nPos = 0;
	nPos = nDiv * (nStad * nX_Y_Z)  + nNmg;


	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02d\tPos : %02d\t", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos);
	//printf("nAxis : %02d\tnStad : %02d\tnX_Y_Z : %02d\t nDiv : %02d\tnNmg : %02d\tPos : %02d\t\n", threadIdx.x, nStad, nX_Y_Z, nDiv, nNmg, nPos);

	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02d\tPos : %02d\tnAxis : %02d\tnStad : %02d\tnX_Y_Z : %02d\t nDiv : %02d\tnNmg : %02d\tPos : %02d\t\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos, threadIdx.x, nStad, nX_Y_Z, nDiv, nNmg, nPos);

	//return;

	//printf("threadIdx.x : %02d\ttid : %02ld\n", threadIdx.x, tid);
	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\ttid : %02d\tPos : %03d\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos);
	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02ld\n", blockIdx.x, blockDim.x, threadIdx.x, tid);
	//return;


	int nOutCnt = 0;
	int nStartIdx = -1, nEndIdx =-1;
	int nIdx = 0;
	for(nIdx = 0 ; nIdx < nX_Y_Z ; nIdx++)
	{
		if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
		{
			nStartIdx = nIdx * nStad;
			break;
		}
		else
			nOutCnt++;
	}

	if(nX_Y_Z != nOutCnt)
	{
		for(nIdx = nX_Y_Z-1 ; nIdx >=0  ; nIdx--)
		{
			if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
			{
				nEndIdx = nIdx * nStad;
				break;
			}
		}
	}

	if(nStartIdx != -1 && nEndIdx != -1)
	{
		//printf("S : %d\t, E : %d\n", nStartIdx, nEndIdx );
		for(unsigned int s = nStartIdx; s <= nEndIdx ; s= s + nStad)
		{
			bResult[nPos + s] = 1;
		}
	}
}



void CGPUSolid::CUDA_SolidVoxelizationXYZ(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelResultTotal)
{
	const unsigned int unMaxBlockCnt = m_nMaxBlockSize;

	const unsigned int unOneThreadSizeX = m_nXFileVoxCnt;
	const unsigned int unOneStepBlcokCntX  = m_nYFileVoxCnt * m_nZFileVoxCnt;

	const unsigned int unOneThreadSizeY = m_nYFileVoxCnt;
	const unsigned int unOneStepBlcokCntY  = m_nXFileVoxCnt * m_nZFileVoxCnt;

	const unsigned int unOneThreadSizeZ = m_nZFileVoxCnt;
	const unsigned int unOneStepBlcokCntZ  = m_nXFileVoxCnt * m_nYFileVoxCnt;


	//! �� �ະ�� ���� ū ����� �������� ó���Ѵ�.
	unsigned int unMaxBlockSize = 0;
	unMaxBlockSize = unOneStepBlcokCntX;
	if(unMaxBlockSize < unOneStepBlcokCntY)
	{
		unMaxBlockSize = unOneStepBlcokCntY;
		printf( "Big Y \n" );
	}
	if(unMaxBlockSize < unOneStepBlcokCntZ)
	{
		unMaxBlockSize = unOneStepBlcokCntZ;
		printf( "Big Z \n" );
	}

	//! ����� �ִ� �ִ� ��� ���� ������ �ִ� ��ϼ���ŭ�� ó���ϱ�����
	unsigned int unOneStepBlcokCnt  = unMaxBlockSize;
	if(unOneStepBlcokCnt > unMaxBlockCnt)
		unOneStepBlcokCnt = unMaxBlockCnt;



	////////////////////////////////////////////////////////////////////////////////////
	//! �޸� �Ҵ�
	bool *cubVoxelState;
	bool *cubVoxelResultX;
	bool *cubVoxelResultY;
	bool *cubVoxelResultZ;

	if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	{
		printf( "Error!  VoxelState Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultX, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! X VoxelResult Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultY, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! X VoxelResult Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultZ, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! X VoxelResult Malloc \n" );
	}
	hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	////////////////////////////////////////////////////////////////////////////////////


	//���� ó���� ����� ����
	unsigned int unBlockStep = 0;
	////////////////////////////////////////////////////////////////////////////////////
	// ! X, Y, Z Inner Voxel ����!!
	while (unBlockStep <  unMaxBlockSize)
	{
		if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
			unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;

		UseCheckXYZ<<<unOneStepBlcokCnt,3>>>(cubVoxelState, cubVoxelResultX, cubVoxelResultY, cubVoxelResultZ, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! BlockStep : %d \n", unBlockStep);
		}
		unBlockStep += unOneStepBlcokCnt;
	}


	printf( "\n\n\n");
	
	
	// �ӽ�
	//hipMemcpy(bVoxelResultTotal, cubVoxelResultZ, unVoxelSize, hipMemcpyDeviceToHost);

	//! �ʿ� ���� �޸𸮴� ������ ��Ż ������ �޸� �Ҵ����ش�.
	hipFree(cubVoxelState);


	//return ;

	////////////////////////////////////////////////////////////////////////////////////
	//! Total ������ ������ش�
	unsigned int unOneStep  = m_nMaxBlockSize * m_nMaxThreadSize;
	if(unOneStep > unVoxelSize)
		unOneStep = unVoxelSize;

	unsigned int unIdx = 0;
	unsigned int unStep = 0;
	while (unStep < unVoxelSize)
	{
		unsigned int unCpySize = 0;
		//! ����̽����� �޸� ī��
		if(unVoxelSize - unStep < unOneStep)
			unCpySize = unVoxelSize - unStep;
		else
			unCpySize = unOneStep;

		bool *cubVoxelTotalResult;
		if ( hipSuccess != hipMalloc(&cubVoxelTotalResult, unCpySize*sizeof(bool)))
		{
			printf( "Error!  VoxelTotal Malloc \n" );
		}
		MergeXYZBig<<<m_nMaxBlockSize,m_nMaxThreadSize>>>(cubVoxelResultX, cubVoxelResultY, cubVoxelResultZ, cubVoxelTotalResult, unCpySize);

		hipMemcpy(bVoxelResultTotal + (unOneStep*unIdx), cubVoxelTotalResult, unCpySize*sizeof(bool), hipMemcpyDeviceToHost);
		hipFree(cubVoxelTotalResult);

		unStep += unCpySize;
		unIdx += 1;
	}
	
	printf( "\n\n\n");

	
	hipFree(cubVoxelResultX);
	hipFree(cubVoxelResultY);
	hipFree(cubVoxelResultZ);
	

}



__global__ void UseCheckXYZBlock(bool *bValue, bool *bResultX, bool *bResultY, bool *bResultZ, int nX, int nY, int nZ, int unBlockStep)
{ 
	unsigned int  tid = blockDim.x * blockIdx.x + threadIdx.x;// + unStep;

	

	//! ��Ϻ� 3�� ����� ���� ó�� �ϱ� ����
	int nAxis = blockIdx.x % 3;
	int nAxisDiv = blockIdx.x / 3;

	////////////////////////////////////////////////
	//! ��Ϻ��� ���� ���ƾ��ϱ⶧�� blockDim.x = 32 ��... ��������
	tid = tid - (blockDim.x * blockIdx.x) + (nAxisDiv * blockDim.x);


	int nDiv = 0;
	int nNmg = 0;
	//! �� �ະ�� ���� ù IDx + ?? �� ����..?
	int nStad = 0;
	//! ��� �� ���� ������ �־���
	int nX_Y_Z = 0;
	bool *bResult = NULL;

	if(nAxis == 0) // X��
	{
		//nStad = nX;
		nStad = 1;
		nX_Y_Z = nX;
		bResult = bResultX;

		if(nY * nZ <= tid)
			return;
	}
	else if(nAxis == 1) // Y��
	{
		nStad = nX;
		nX_Y_Z = nY;
		bResult = bResultY;

		if(nX * nZ <= tid)
			return;
	}
	else //if(nAxis == 2) Z��
	{
		nStad = nX*nY;
		nX_Y_Z = nZ;
		bResult = bResultZ;
		
		if(nX * nY <= tid)
			return;

	}

	//! �Ʒ��� ���..
	if((unBlockStep + tid) > 0)
	{
		nDiv = (unBlockStep + tid) / nStad;
		nNmg = (unBlockStep + tid) % nStad;
	}



	int nPos = 0;
	nPos = nDiv * (nStad * nX_Y_Z)  + nNmg;

	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02d\tPos : %02d\t\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos);
	//printf("nAxis : %02d\tnStad : %02d\tnX_Y_Z : %02d\t nDiv : %02d\tnNmg : %02d\tPos : %02d\t\n", nAxis, nStad, nX_Y_Z, nDiv, nNmg, nPos);
	//printf("blockIdx.x : %02d\tblockDim.x : %02d\tthreadIdx.x : %02d\t tid : %02d\tPos : %02d\tnAxis : %02d\tnStad : %02d\tnX_Y_Z : %02d\t nDiv : %02d\tnNmg : %02d\tPos : %02d\t\n", blockIdx.x, blockDim.x, threadIdx.x, tid, nPos, nAxis, nStad, nX_Y_Z, nDiv, nNmg, nPos);

	//return;

	int nOutCnt = 0;
	int nStartIdx = -1, nEndIdx =-1;
	int nIdx = 0;
	for(nIdx = 0 ; nIdx < nX_Y_Z ; nIdx++)
	{
		if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
		{
			nStartIdx = nIdx * nStad;
			break;
		}
		else
			nOutCnt++;
	}

	if(nX_Y_Z != nOutCnt)
	{
		for(nIdx = nX_Y_Z-1 ; nIdx >=0  ; nIdx--)
		{
			if(bValue[nPos + (nIdx * nStad)])	// ù �����͸� ã�´�.
			{
				nEndIdx = nIdx * nStad;
				break;
			}
		}
	}

	if(nStartIdx != -1 && nEndIdx != -1)
	{
		//printf("S : %d\t, E : %d\n", nStartIdx, nEndIdx );
		for(unsigned int s = nStartIdx; s <= nEndIdx ; s= s + nStad)
		{
			bResult[nPos + s] = 1;
		}
	}
}



void CGPUSolid::CUDA_SolidVoxelizationXYZBlock(unsigned int  unVoxelSize, bool *bVoxelState, bool *bVoxelResultTotal)
{
	const unsigned int unMaxBlockCnt = m_nMaxBlockSize;

	const unsigned int unOneThreadSizeX = m_nXFileVoxCnt;
	const unsigned int unOneStepBlcokCntX  = m_nYFileVoxCnt * m_nZFileVoxCnt;

	const unsigned int unOneThreadSizeY = m_nYFileVoxCnt;
	const unsigned int unOneStepBlcokCntY  = m_nXFileVoxCnt * m_nZFileVoxCnt;

	const unsigned int unOneThreadSizeZ = m_nZFileVoxCnt;
	const unsigned int unOneStepBlcokCntZ  = m_nXFileVoxCnt * m_nYFileVoxCnt;


	//! �� �ະ�� ���� ū ����� �������� ó���Ѵ�.
	unsigned int unMaxBlockSize = 0;
	unMaxBlockSize = unOneStepBlcokCntX;
	if(unMaxBlockSize < unOneStepBlcokCntY)
	{
		unMaxBlockSize = unOneStepBlcokCntY;
		printf( "Big Y \n" );
	}
	if(unMaxBlockSize < unOneStepBlcokCntZ)
	{
		unMaxBlockSize = unOneStepBlcokCntZ;
		printf( "Big Z \n" );
	}
	
	//! ���� 3���ϱ�!
	unMaxBlockSize = ((unMaxBlockSize / 32) + 1) * 3;
	//unMaxBlockSize = ((unMaxBlockSize / 32) + 1) * 3;

	//! ����� �ִ� �ִ� ��� ���� ������ �ִ� ��ϼ���ŭ�� ó���ϱ����� -> +1�� �������� ó���� ���־���ؼ�..
	unsigned int unOneStepBlcokCnt  = unMaxBlockSize;
	if(unOneStepBlcokCnt > unMaxBlockCnt)
		unOneStepBlcokCnt = unMaxBlockCnt;



	////////////////////////////////////////////////////////////////////////////////////
	//! �޸� �Ҵ�
	bool *cubVoxelState;
	bool *cubVoxelResultX;
	bool *cubVoxelResultY;
	bool *cubVoxelResultZ;

	if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	{
		printf( "Error!  VoxelState Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultX, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! X VoxelResult Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultY, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Y VoxelResult Malloc \n" );
	}
	if ( hipSuccess != hipMalloc(&cubVoxelResultZ, unVoxelSize*sizeof(bool)))
	{
		printf( "Error! Z VoxelResult Malloc \n" );
	}
	hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	////////////////////////////////////////////////////////////////////////////////////
	//���� ó���� ����� ����
	unsigned int unBlockStep = 0;
	////////////////////////////////////////////////////////////////////////////////////
	// ! X, Y, Z Inner Voxel ����!!
	while (unBlockStep <  unMaxBlockSize)
	{
		if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
			unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;

		UseCheckXYZBlock<<<unOneStepBlcokCnt,32>>>(cubVoxelState, cubVoxelResultX, cubVoxelResultY, cubVoxelResultZ, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! BlockStep : %d \n", unBlockStep);
		}
		unBlockStep += unOneStepBlcokCnt;
	}


	// �ӽ�
//	hipMemcpy(bVoxelResultTotal, cubVoxelResultZ, unVoxelSize, hipMemcpyDeviceToHost);


	//! �ʿ� ���� �޸𸮴� ������ ��Ż ������ �޸� �Ҵ����ش�.
	hipFree(cubVoxelState);


	//return ;

	////////////////////////////////////////////////////////////////////////////////////
	//! Total ������ ������ش�
	unsigned int unOneStep  = m_nMaxBlockSize * m_nMaxThreadSize;
	if(unOneStep > unVoxelSize)
		unOneStep = unVoxelSize;

	unsigned int unIdx = 0;
	unsigned int unStep = 0;
	while (unStep < unVoxelSize)
	{
		unsigned int unCpySize = 0;
		//! ����̽����� �޸� ī��
		if(unVoxelSize - unStep < unOneStep)
			unCpySize = unVoxelSize - unStep;
		else
			unCpySize = unOneStep;

		bool *cubVoxelTotalResult = NULL;
		if ( hipSuccess != hipMalloc(&cubVoxelTotalResult, unCpySize*sizeof(bool)))
		{
			printf( "Error!  VoxelTotal Malloc \n" );
		}
		MergeXYZBig<<<m_nMaxBlockSize,m_nMaxThreadSize>>>(cubVoxelResultX, cubVoxelResultY, cubVoxelResultZ, cubVoxelTotalResult, unCpySize);

		hipMemcpy(bVoxelResultTotal + (unOneStep*unIdx), cubVoxelTotalResult, unCpySize*sizeof(bool), hipMemcpyDeviceToHost);
		hipFree(cubVoxelTotalResult);

		unStep += unCpySize;
		unIdx += 1;
	}
	
	
	hipFree(cubVoxelResultX);
	hipFree(cubVoxelResultY);
	hipFree(cubVoxelResultZ);
	

}

