#include "hip/hip_runtime.h"
#include "GPUSurface.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>


#define SIZE 1024

CGPUSurface::CGPUSurface(void)
{
	//! �ػ� ������ ����
	g_nDivideCnt = 0;
	//! ���õ� ��
	g_nSelectAxis = 0;
	//! ���� ũ�� �ε�
	g_fRectSize = 0.0;
	////////////////////////////////////////
	//Object ����ȭ ����
	m_nUseVertex = 0;
	memset(vertexes, NULL, sizeof(sVertexCuda) * dfMAX_VERTEX_CNT);

	m_nUsePart = 0;
	memset(parts, NULL, sizeof(sPartCuda) * dfMAX_PART_CNT);

	m_fObjMaxX = 0.0;
	m_fObjMaxY = 0.0;
	m_fObjMaxZ = 0.0;
	m_fObjMinX = 0.0;
	m_fObjMinY = 0.0;
	m_fObjMinZ = 0.0;

	//////////////////////////////////////////
	// etc
	m_nXFileVoxCnt = 0;
	m_nYFileVoxCnt = 0;
	m_nZFileVoxCnt = 0;

}

CGPUSurface::~CGPUSurface(void)
{

}



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//! ǥ�� ����

void CGPUSurface::SetGlobalValue(int		nDivideCnt, int		nSelectAxis, float	fRectSize)
{
	g_nDivideCnt = nDivideCnt;
	g_nSelectAxis = nSelectAxis;
	g_fRectSize = fRectSize;

}


void CGPUSurface::SetObjectValue(float fObjMaxX, float fObjMaxY,float fObjMaxZ,	float fObjMinX,	float fObjMinY,	float fObjMinZ,
		int nVretexesSize , char *pVertex, int nPartSize, char *pParts)
{
	m_fObjMaxX = fObjMaxX;
	m_fObjMaxY = fObjMaxY;
	m_fObjMaxZ = fObjMaxZ;
	m_fObjMinX = fObjMinX;
	m_fObjMinY = fObjMinY;
	m_fObjMinZ = fObjMinZ;

	m_nUseVertex = nVretexesSize;
	memcpy(vertexes, pVertex, sizeof(sVertexCuda) * nVretexesSize);

	m_nUsePart = nPartSize;
	memcpy(parts, pParts, sizeof(sPartCuda) * nPartSize);
}


void CGPUSurface::SetGlobalEtc(unsigned long long nXFileVoxCnt,	unsigned long long nYFileVoxCnt,	unsigned long long nZFileVoxCnt)
{
	m_nXFileVoxCnt = nXFileVoxCnt;
	m_nYFileVoxCnt = nYFileVoxCnt;
	m_nZFileVoxCnt = nZFileVoxCnt;
}



__global__ void CreateFaceArea(sFaceCuda* sFaceData, sVertexCuda* sVertexData, sPartDataCuda*	cusPartDataResult)
{ 
	int tid = blockIdx.x * blockDim.x+ threadIdx.x;	
	
	printf("blockIdx : %d\t blockDim : %d\t threadIdx : %d\t tid : %d\n", blockIdx.x, blockDim.x, threadIdx.x, tid);

	for (int j=0,jn= sFaceData[tid].n; j<jn; ++j) 
	{
		int &v  = sFaceData[tid].v[j];
		//if (0 < v && v <= vt_size) 
		{
			sVertexCuda &vt = sVertexData[v - 1];

			//triverts[j][0] = vt.x;
			//triverts[j][1] = vt.y;
			//triverts[j][2] = vt.z;

			//if(PartData.fMaxX < vt.x)
			//	PartData.fMaxX = vt.x;
			//if(PartData.fMaxY < vt.y)
			//	PartData.fMaxY = vt.y;
			//if(PartData.fMaxZ < vt.z)
			//	PartData.fMaxZ = vt.z;

			//if(PartData.fMinX > vt.x)
			//	PartData.fMinX = vt.x;
			//if(PartData.fMinY > vt.y)
			//	PartData.fMinY = vt.y;
			//if(PartData.fMinZ > vt.z)
			//	PartData.fMinZ = vt.z;

			//nPartCnt++;
		}

		//TRACE("\tv:%d\t", v);

	}

	sFaceData[tid].v[0];
	sFaceData[tid].v[1];
	sFaceData[tid].v[2];
	sFaceData[tid].v[3];
	sFaceData[tid].v[4];


}

void CGPUSurface::SetSurfaceVoxelization()
{
	m_nUsePart;
	parts[0].nUseFaceCnt;

	for(int nPart = 0; nPart < m_nUsePart; nPart++)
	{
		sFaceCuda* cusFaceData = NULL;
		sVertexCuda* cusVertexData = NULL;

		sPartDataCuda*	cusPartDataCuda = NULL;

		if ( hipSuccess != hipMalloc(&cusFaceData, parts[nPart].nUseFaceCnt*sizeof(sFaceCuda) ) )
		{
			printf( "Error! Face MemCpy \n" );
		}
		if ( hipSuccess != hipMalloc(&cusVertexData, m_nUseVertex*sizeof(sVertexCuda) ) )
		{
			printf( "Error! Vertex MemCpy \n" );
		}
		if ( hipSuccess != hipMalloc(&cusPartDataCuda, parts[nPart].nUseFaceCnt*sizeof(sPartDataCuda) ) )
		{
			printf( "Error! Face MemCpy \n" );
		}
		

		hipMemcpy(cusFaceData, &parts[nPart], parts[nPart].nUseFaceCnt*sizeof(sFaceCuda), hipMemcpyHostToDevice);
		hipMemcpy(cusVertexData, &vertexes, m_nUseVertex*sizeof(sVertexCuda), hipMemcpyHostToDevice);

		CreateFaceArea<<<parts[nPart].nUseFaceCnt,1>>>(cusFaceData, cusVertexData, cusPartDataCuda);
		if ( hipSuccess != hipGetLastError() )
		{
			printf( "Error! Kernel : %d \n");
		}

		hipFree(cusFaceData);
		hipFree(cusVertexData);
	}



	//const unsigned int unOneThreadSize = m_nYFileVoxCnt;
	//const unsigned int unMaxBlockCnt = m_nMaxBlockSize;
	//unsigned int unOneStepBlcokCnt  = m_nXFileVoxCnt * m_nZFileVoxCnt;
	//if(unOneStepBlcokCnt > unMaxBlockCnt)
	//{
	//	unOneStepBlcokCnt = unMaxBlockCnt;
	//}
	//		
	//const unsigned int unMaxBlockSize = unVoxelSize / m_nYFileVoxCnt;

	//unsigned int unBlockStep = 0;

	//bool *cubVoxelState;
	//bool *cubVoxelResultSize;

	//if ( hipSuccess != hipMalloc(&cubVoxelState, unVoxelSize*sizeof(bool)))
	//{
	//	printf( "Error! Y VoxelState Malloc \n" );
	//}
	//if ( hipSuccess != hipMalloc(&cubVoxelResultSize, unVoxelSize*sizeof(bool)))
	//{
	//	printf( "Error! Y VoxelResult Malloc \n" );
	//}

	////! ����̽����� �޸� ī��
	//hipMemcpy(cubVoxelState, bVoxelState, unVoxelSize*sizeof(bool), hipMemcpyHostToDevice);

	//while (unBlockStep <  unMaxBlockSize)
	//{
	//	if(unOneStepBlcokCnt > unMaxBlockSize - unBlockStep)
	//		unOneStepBlcokCnt = unMaxBlockSize - unBlockStep;
	//	CreateFaceArea<<<unOneStepBlcokCnt,1>>>(cubVoxelState, cubVoxelResultSize, m_nXFileVoxCnt, m_nYFileVoxCnt, m_nZFileVoxCnt, unBlockStep);
	//	if ( hipSuccess != hipGetLastError() )
	//	{
	//		printf( "Error! Y BlockStep : %d \n", unBlockStep);
	//	}
	//	unBlockStep += unOneStepBlcokCnt;
	//	
	//}
	//
	//if ( hipSuccess != hipMemcpy(bVoxelYResult, cubVoxelResultSize, unVoxelSize*sizeof(bool), hipMemcpyDeviceToHost))
	//{
	//	printf( "Error! Y VoxelState Memcpy \n" );
	//}

	//hipFree(cubVoxelState);
	//hipFree(cubVoxelResultSize);


}